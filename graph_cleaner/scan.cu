#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 
#define thread_limit 256 
#define block_limit 1024 

#define GPU_COWORKER 1 
#define warp_thd 256 
#define block_thd 256
graph * mygraph;
//#define GPU_NUM 4 
//int     NUM_ONE_TASK;

__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
//	index_t*	degree,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ block_thd;
	int i = threadIdx.x% block_thd;
	index_t mycount=0;
//	__shared__ int cache[256];
//	__shared__ int offset[256];
	__shared__ index_t local[block_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];
		
		index_t temp;
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/block_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = block_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + block_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
//			bot = bot*k;
//			top = top*k;
			bot = bot*m/block_thd;
			top = top*m/block_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += block_thd;
		
		}
		tid += GPU_COWORKER * gridDim.x*blockDim.x/ block_thd;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
//	index_t*	degree,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[warp_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
//			bot = bot*k;
//			top = top*k;
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += GPU_COWORKER* blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();

}

//----------------------------------------------------------------------------------------

__global__ void classify_kernel	//step 1: classify the edge list into different arrays
(	vertex_t* adj_list,
	vertex_t* head_list,
	index_t* begin,
	index_t  N,		//inputs
	index_t* small_num,
	index_t* mid_num,
	index_t* large_num
	//outputs: small/large head, adjacent, and number by thread
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space
	index_t small_offset=0;
	index_t mid_offset=0;
	index_t large_offset=0;
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit){
				small_offset++;
			}
			else if(n<block_limit){	//could be more then 2 catigories
//			else{
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				large_offset++;
			}
		}
	}
	small_num[tid] = small_offset;
	mid_num[tid]   = mid_offset;
	large_num[tid] = large_offset;

}

__global__ void prefix_kernel_1	//this prefix scan function could be easier for data size is always 256*256
(	
 	index_t*	data,
	index_t*	block_offset
)
{
		
	//step 1: each block do prefix sum inside
	int tid = threadIdx.x +blockIdx.x*blockDim.x;

	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = data[tid];
	__syncthreads();

	index_t val=0;
	for(int i=0; i<=threadIdx.x; i++){
		val += temp_in[i];
	}


	__syncthreads();
	
	if(threadIdx.x==255){
		block_offset[blockIdx.x] = val;
		
	}
	data[tid] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_2	
(	
	index_t*	block_offset
)
{
	//step 2: collect each block's offset and do prefix for this set
	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = block_offset[threadIdx.x];
	__syncthreads();
	index_t val=0;
	for(int i=0; i<threadIdx.x; i++){
		val += temp_in[i];
	}
//		val = temp_in[threadIdx.x];
	block_offset[threadIdx.x] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_3	
(	
	index_t*	data,
	index_t*	block_offset
)
{
	//step 3: update by adding block offset
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	index_t val = data[tid];
	index_t offset = block_offset[blockIdx.x];
	val += offset;

	data[tid] = val;
	__syncthreads();
}

__global__ void collect_kernel
(	vertex_t* 	adj_list,
	vertex_t* 	head_list,
	index_t* 	begin,
	index_t	N,
	index_t* 	small_num,
	index_t* 	mid_num,
	index_t* 	large_num,
	index_t 	N1,
	index_t	N2,
	vertex_t*	dest_head,
	vertex_t*	dest_adj
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space


	index_t thd_base_small = 0;
	index_t thd_base_mid   = N1;
	index_t thd_base_large = N1+N2;
	if(tid!=0){
		thd_base_small = small_num[tid-1];
		thd_base_mid   = N1 + mid_num[tid-1];
		thd_base_large = N1 + N2 + large_num[tid-1];
	}
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	index_t small_offset = thd_base_small;
	index_t mid_offset   = thd_base_mid;
	index_t large_offset = thd_base_large;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit){
				dest_head[small_offset] = head;
				dest_adj [small_offset] = adj;
				small_offset++;
			}
			else if(n<block_limit){	//could be more then 2 catigories
//			else{
				dest_head[mid_offset] = head;
				dest_adj [mid_offset] = adj;
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				dest_head[large_offset] = head;
				dest_adj [large_offset] = adj;
				large_offset++;
			}
		}
	}
}


__global__ void reduce_kernel2(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void* part_scan(void * data){

	index_t thd_count=0;	
	int GPU_id = *(int*)data;
	int i = GPU_id;
//	cout<<"GPU id = "<<GPU_id<<"\n";
	hipSetDevice(GPU_id);
	H_ERR(hipDeviceSynchronize() );

	vertex_t*	dev_adj;
	vertex_t*	dev_head;
//	index_t*	dev_degree;
	index_t*	dev_begin;
	index_t*	dev_count;

	index_t partEdgeCount = mygraph->partEdgeCount[i];
	vertex_t vert_count = mygraph->vert_count;
	vertex_t* partAdj = mygraph->partAdj[i];
	vertex_t* partHead= mygraph->partHead[i];
//	index_t* partDegree = mygraph->partDegree[i];
	index_t* partBegin  = mygraph->partBegin[i];
	index_t* count    = mygraph->count;

	H_ERR(hipMalloc(&dev_adj, partEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_head, partEdgeCount*sizeof(vertex_t)) );
//	H_ERR(hipMalloc(&dev_degree, vert_count*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_begin,  vert_count*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

		
	
	index_t* block_offset;
	H_ERR(hipMalloc(&block_offset, max_block*sizeof(index_t)) );
	
	H_ERR(hipMemcpy(dev_adj,    partAdj, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_head,   partHead, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
//	H_ERR(hipMemcpy(dev_degree, partDegree, vert_count*sizeof(index_t),  hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  partBegin,  vert_count*sizeof(index_t),  hipMemcpyHostToDevice) );

	for(int j=0; j<PART_NUM; j++){	
		index_t totalEdgeCount = mygraph->partEdgeCount[j];
		vertex_t* 	head = mygraph->partHead[j];
		vertex_t* 	adj  = mygraph->partAdj[j];
//		index_t* 	degree = mygraph->partDegree[j];
		vertex_t*	classified_head;
		vertex_t*	classified_adj;
		
		index_t*	small_num;
		index_t*	mid_num;
		index_t*	large_num;

		vertex_t*	src_head;
		vertex_t*	src_adj;
//		index_t*	src_degree;
		
		H_ERR(hipMalloc(&small_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&mid_num,   max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&large_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&src_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&src_adj,  totalEdgeCount*sizeof(vertex_t)) );
//		H_ERR(hipMalloc(&src_degree,  vert_count*sizeof(index_t)) );
		
		H_ERR(hipMemcpy(src_adj,    adj, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(src_head,   head, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
//		H_ERR(hipMemcpy(src_degree, degree, vert_count*sizeof(index_t), hipMemcpyHostToDevice) );
		
		H_ERR(hipMalloc(&classified_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&classified_adj,  totalEdgeCount*sizeof(vertex_t)) );
		//

	double time1=wtime();
		H_ERR(hipDeviceSynchronize() );

		
		classify_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					dev_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num
					);
		H_ERR(hipDeviceSynchronize() );

		//test for prefix sum

		prefix_kernel_1 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

		prefix_kernel_1 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		
		prefix_kernel_1 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

	index_t N1,N2,N3;	
		H_ERR(hipMemcpy(&N1 ,  &small_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N2 , &mid_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N3 ,  &large_num[65535]   , sizeof(index_t), hipMemcpyDeviceToHost) );

		H_ERR(hipDeviceSynchronize() );
	//	cout<<"N1 = "<<N1<<"\n";
	//	cout<<"N2 = "<<N2<<"\n";
	//	cout<<"N3 = "<<N3<<"\n";
		
		collect_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					dev_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num,
					N1,
					N2,
					classified_head,
					classified_adj
					);
		H_ERR(hipDeviceSynchronize() );


	//double time2=wtime();


		warp_binary_kernel<<<max_block,warp_thd>>>
		(	classified_head,
			classified_adj,
			dev_adj,
//			dev_degree,
			dev_begin,
			0,
			N1,
			dev_count
		);
		H_ERR(hipDeviceSynchronize() );

	//double time3 = wtime();
	/*
	//	int* 	large_head = &(classified_head[N1]);
	//	int*	large_adj  = &(classified_adj[N1]);
		block_binary_kernel<<<max_block,block_thd>>>
		(	classified_head,
			classified_adj,
			dev_adj,
//			dev_degree,
			dev_begin,
			N1+GPU_id*256,
			N1+N2,
			dev_count
		);
		H_ERR(hipDeviceSynchronize() );
	*/
	//double time4 = wtime();
		block_binary_kernel<<<max_block,max_thd>>>
		(	classified_head,
			classified_adj,
			//dev_head,
			//dev_adj,

			dev_adj,
//			dev_degree,
			dev_begin,
			N1,
	//		0 + GPU_id*256,
			totalEdgeCount,
			dev_count
		);
		H_ERR(hipDeviceSynchronize() );
		
		reduce_kernel2 <<<1,1>>>(dev_count);
		H_ERR(hipDeviceSynchronize() );
		
		H_ERR(hipMemcpy(&count[i], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
		thd_count += count[i];
		
		H_ERR(hipFree(small_num) );
		H_ERR(hipFree(large_num) );
		H_ERR(hipFree(classified_head) );
		H_ERR(hipFree(classified_adj) );
		H_ERR(hipFree(src_head) );
		H_ERR(hipFree(src_adj) );
//		H_ERR(hipFree(src_degree) );
		cout<<"GPU "<<i<<" part "<<j<<"\n";
	}

	count[i] = thd_count;
	cout<<"gpu binary count="<<count[i]<<"\n";
	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_head) );
//	H_ERR(hipFree(dev_degree) );
	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(block_offset) );
	H_ERR(hipFree(dev_count) );
	return NULL;	
}


