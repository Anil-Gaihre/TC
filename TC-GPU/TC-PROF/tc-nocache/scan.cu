#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 
#define thread_limit 256 
#define block_limit 1024 

#define GPU_COWORKER 1 
graph * mygraph;
long	total_count;

__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,

	long int*	counter_1,
	long int*	counter_2,

	index_t*	count
)
{
	int p = threadIdx.x/32;
	long counter1=0;
	long counter2=0;
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];



		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
//		local[i]=a[i*m/max_thd];	
		__syncthreads();

	counter1 += 8;
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
	counter1++;
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
/*
			while(top>bot+1){
				
		__syncthreads();
				warp_path[3*p]=0;
				warp_path[3*p+1]=0;
				warp_path[3*p+2]=0;
		__syncthreads();

				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + max_thd;
					warp_path[3*p]=1;
				}
				if(X<Y){
					top = r;
					warp_path[3*p+1]=1;
				}
				if(X>Y){
					bot = r;
					warp_path[3*p+2]=1;
				}

				int k=0;
				if(warp_path[3*p]!=0){
					k++;
				}
				if(warp_path[3*p+1]!=0){
					k++;
				}
				if(warp_path[3*p+2]!=0){
					k++;
				}
		counter2 +=k;

			}
*/
			//phase 2
//			bot = bot*m/max_thd;
//			top = top*m/max_thd -1;
			bot = 0;
			top = m-1;
			while(top>=bot){
		

				r = (top+bot)/2;
				Y = a[r];
	counter1++;
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
				
			
			}
			j += max_thd;
		
		}
		tid += GPU_COWORKER * gridDim.x*blockDim.x/ max_thd;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
	counter_1[blockDim.x*blockIdx.x+threadIdx.x]+=counter1;
	counter_2[blockDim.x*blockIdx.x+threadIdx.x]+=counter2;
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,

	long int*	counter_1,
	long int*	counter_2,

	index_t*	count
)
{
	long counter1=0;
	long counter2=0;
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
//		local[p*32+i]=a[i*m/32];	
	counter1+=8;
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
	counter1++;
			vertex_t Y;

			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
/*
			while(top>bot+1){
		__syncthreads();
				warp_path[3*p]=0;
				warp_path[3*p+1]=0;
				warp_path[3*p+2]=0;
		__syncthreads();
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
					warp_path[3*p]=1;
				}
				if(X<Y){
					top = r;
					warp_path[3*p+1]=1;
				}
				if(X>Y){
					bot = r;
					warp_path[3*p+2]=1;
				}
				int k=0;
				if(warp_path[3*p]!=0){
					k++;
				}
				if(warp_path[3*p+1]!=0){
					k++;
				}
				if(warp_path[3*p+2]!=0){
					k++;
				}
		counter2 +=k;
			}
*/
			//phase 2
			bot = 0;
			top = m -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
	counter1++;
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += GPU_COWORKER* blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();
	counter_1[blockDim.x*blockIdx.x+threadIdx.x]=counter1;
	counter_2[blockDim.x*blockIdx.x+threadIdx.x]=counter2;

}

//----------------------------------------------------------------------------------------

__global__ void classify_kernel	//step 1: classify the edge list into different arrays
(	vertex_t* adj_list,
	vertex_t* head_list,
	index_t* begin,
	index_t  N,		//inputs
	index_t* small_num,
	index_t* mid_num,
	index_t* large_num
	//outputs: small/large head, adjacent, and number by thread
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space
	index_t small_offset=0;
	index_t mid_offset=0;
	index_t large_offset=0;
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
//			else{
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				large_offset++;
			}
		}
	}
	small_num[tid] = small_offset;
	mid_num[tid]   = mid_offset;
	large_num[tid] = large_offset;

}

__global__ void prefix_kernel_1	//this prefix scan function could be easier for data size is always 256*256
(	
 	index_t*	data,
	index_t*	block_offset
)
{
		
	//step 1: each block do prefix sum inside
	int tid = threadIdx.x +blockIdx.x*blockDim.x;

	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = data[tid];
	__syncthreads();

	index_t val=0;
	for(int i=0; i<=threadIdx.x; i++){
		val += temp_in[i];
	}


	__syncthreads();
	
	if(threadIdx.x==255){
		block_offset[blockIdx.x] = val;
		
	}
	data[tid] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_2	
(	
	index_t*	block_offset
)
{
	//step 2: collect each block's offset and do prefix for this set
	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = block_offset[threadIdx.x];
	__syncthreads();
	index_t val=0;
	for(int i=0; i<threadIdx.x; i++){
		val += temp_in[i];
	}
//		val = temp_in[threadIdx.x];
	block_offset[threadIdx.x] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_3	
(	
	index_t*	data,
	index_t*	block_offset
)
{
	//step 3: update by adding block offset
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	index_t val = data[tid];
	index_t offset = block_offset[blockIdx.x];
	val += offset;

	data[tid] = val;
	__syncthreads();
}

__global__ void collect_kernel
(	vertex_t* 	adj_list,
	vertex_t* 	head_list,
	index_t* 	begin,
	index_t	N,
	index_t* 	small_num,
	index_t* 	mid_num,
	index_t* 	large_num,
	index_t 	N1,
	index_t	N2,
	vertex_t*	dest_head,
	vertex_t*	dest_adj
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space


	index_t thd_base_small = 0;
	index_t thd_base_mid   = N1;
	index_t thd_base_large = N1+N2;
	if(tid!=0){
		thd_base_small = small_num[tid-1];
		thd_base_mid   = N1 + mid_num[tid-1];
		thd_base_large = N1 + N2 + large_num[tid-1];
	}
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	index_t small_offset = thd_base_small;
	index_t mid_offset   = thd_base_mid;
	index_t large_offset = thd_base_large;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				dest_head[small_offset] = head;
				dest_adj [small_offset] = adj;
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
//			else{
				dest_head[mid_offset] = head;
				dest_adj [mid_offset] = adj;
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				dest_head[large_offset] = head;
				dest_adj [large_offset] = adj;
				large_offset++;
			}
		}
	}
}

__global__ void reduce_kernel_count(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block*max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

__global__ void reduce_kernel(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void* part_scan(void * data){

	index_t thd_count=0;	
	int GPU_id = *(int*)data;
	int i = GPU_id;
//	cout<<"GPU id = "<<GPU_id<<"\n";
	hipSetDevice(GPU_id);
	H_ERR(hipDeviceSynchronize() );

	vertex_t*	dev_adj;
	vertex_t*	dev_head;
	index_t*	dev_begin;
	index_t*	dev_count;

	index_t partEdgeCount = mygraph->partEdgeCount[i];
	vertex_t vert_count = mygraph->vert_count;
	vertex_t* partAdj = mygraph->partAdj[i];
	vertex_t* partHead= mygraph->partHead[i];
//	index_t* partDegree = mygraph->partDegree[i];
	index_t* partBegin  = mygraph->partBegin[i];
	index_t* count    = mygraph->count;

	H_ERR(hipMalloc(&dev_adj, partEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_head, partEdgeCount*sizeof(vertex_t)) );
//	H_ERR(hipMalloc(&dev_degree, vert_count*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

		
	
	index_t* block_offset;
	H_ERR(hipMalloc(&block_offset, max_block*sizeof(index_t)) );
	
	H_ERR(hipMemcpy(dev_adj,    partAdj, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_head,   partHead, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
//	H_ERR(hipMemcpy(dev_degree, partDegree, vert_count*sizeof(index_t),  hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  partBegin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );
	
long 	counter_1_cpu=0;
//long 	counter_2_cpu=0;
long int tmp_counter1,tmp_counter2;
long int*       counter_1;//counter for memory read
long int*       counter_2;//counter for divergence
H_ERR(hipMalloc(&counter_1,    max_thd*max_block*sizeof(long int)) );
H_ERR(hipMalloc(&counter_2,    max_thd*max_block*sizeof(long int)) );

	double time2=wtime();
	for(int j=0; j<PART_NUM; j++){	
		index_t totalEdgeCount = mygraph->partEdgeCount[j];
		vertex_t* 	head = mygraph->partHead[j];
		vertex_t* 	adj  = mygraph->partAdj[j];
		vertex_t*	classified_head;
		vertex_t*	classified_adj;
		
		index_t*	small_num;
		index_t*	mid_num;
		index_t*	large_num;

		vertex_t*	src_head;
		vertex_t*	src_adj;
		
		H_ERR(hipMalloc(&small_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&mid_num,   max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&large_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&src_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&src_adj,  totalEdgeCount*sizeof(vertex_t)) );
		
		H_ERR(hipMemcpy(src_adj,    adj, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(src_head,   head, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		
		H_ERR(hipMalloc(&classified_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&classified_adj,  totalEdgeCount*sizeof(vertex_t)) );
		//

	double time1=wtime();
		H_ERR(hipDeviceSynchronize() );

		
		classify_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					dev_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num
					);
		H_ERR(hipDeviceSynchronize() );

		//test for prefix sum

		prefix_kernel_1 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

		prefix_kernel_1 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		
		prefix_kernel_1 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

	index_t N1,N2,N3;	
		H_ERR(hipMemcpy(&N1 ,  &small_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N2 , &mid_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N3 ,  &large_num[65535]   , sizeof(index_t), hipMemcpyDeviceToHost) );

		H_ERR(hipDeviceSynchronize() );
	//	cout<<"N1 = "<<N1<<"\n";
	//	cout<<"N2 = "<<N2<<"\n";
	//	cout<<"N3 = "<<N3<<"\n";
		
		collect_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					dev_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num,
					N1,
					N2,
					classified_head,
					classified_adj
					);
		H_ERR(hipDeviceSynchronize() );




		warp_binary_kernel<<<max_block,max_thd>>>
		(	classified_head,
			classified_adj,
			dev_adj,
			dev_begin,
			0,
			N1,
			
			counter_1,
			counter_2,

			dev_count
		);
		H_ERR(hipDeviceSynchronize() );

		block_binary_kernel<<<max_block,max_thd>>>
		(	classified_head,
			classified_adj,
			dev_adj,
			dev_begin,
			N1,
			N1+N2,
	//		0 + GPU_id*256,
	//		totalEdgeCount,
			
			counter_1,
			counter_2,

			dev_count
		);
		H_ERR(hipDeviceSynchronize() );
		
		reduce_kernel <<<1,1>>>(dev_count);
		H_ERR(hipDeviceSynchronize() );
		
		H_ERR(hipMemcpy(&count[i], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
		thd_count += count[i];
		
		
		reduce_kernel_count <<<1,1>>>(counter_1);
		H_ERR(hipDeviceSynchronize() );
//		reduce_kernel_count <<<1,1>>>(counter_2);
//		H_ERR(hipDeviceSynchronize() );
		//long int tmp_counter1,tmp_counter2;
		H_ERR(hipMemcpy(&tmp_counter1, counter_1, sizeof(long), hipMemcpyDeviceToHost));
//		H_ERR(hipMemcpy(&tmp_counter2, counter_2, sizeof(long), hipMemcpyDeviceToHost));
		counter_1_cpu += tmp_counter1;
//		counter_2_cpu += tmp_counter2;
	
	
		H_ERR(hipFree(small_num) );
		H_ERR(hipFree(large_num) );
		H_ERR(hipFree(classified_head) );
		H_ERR(hipFree(classified_adj) );
		H_ERR(hipFree(src_head) );
		H_ERR(hipFree(src_adj) );
//		H_ERR(hipFree(src_begin) );
		cout<<"GPU "<<i<<" part "<<j<<"\n";
	}
	
	double time4 = wtime();
	count[i] = thd_count;
	cout<<"gpu "<<i<<" binary count="<<count[i]<<"\n";
	cout<<"time = "<<time4-time2<<" seconds"<<endl;

	cout<<"counter for mem_read   = "<<counter_1_cpu<<endl;
//	cout<<"counter for divergence = "<<counter_2_cpu<<endl;
	total_count += counter_1_cpu;

	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_head) );
//	H_ERR(hipFree(dev_degree) );
	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(block_offset) );
	H_ERR(hipFree(dev_count) );
	return NULL;	
}


