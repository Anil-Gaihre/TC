#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 
#define thread_limit 256 
#define block_limit 1024 

#define warp_thd 256 
#define block_thd 256
graph * mygraph;
//#define GPU_NUM 4 
//int     NUM_ONE_TASK;

__global__ void warp_merge_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
//	index_t*	iter_count,
	index_t*	count
)
{
//	index_t iter=0;
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	int i = threadIdx.x%32;
	int p = threadIdx.x/32;
	index_t mycount=0;
	__shared__ index_t local[max_thd];
	__shared__ index_t A_diag[33*8];
	__shared__ index_t B_diag[33*8];	
	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];// degree[A];
		index_t n = begin[B+1]-begin[B];// degree[B];

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
		if(i==0){
			A_diag[p*33+32]=m;
			B_diag[p*33+32]=n;
		}
		index_t index = (m+n)/32*i;
		index_t A_top, A_bottom, B_top, Ai, Bi;
		if(index>m){
			A_top = m;
			B_top = index-m;
		}
		else if(index<=m){
			A_top = index;
			B_top = 0;
		}
		if(index>n){
			A_bottom = index-n;
		}
		else if(index<=n){
			A_bottom = 0;
		}
//partition
		while(1){
			index_t offset=(A_top-A_bottom)/2;
			if(A_top==A_bottom){
				A_diag[p*33+i]=A_top;
				B_diag[p*33+i]=B_top;
				break;
			}

			Ai = A_top - offset;
			Bi = B_top + offset;
			if(offset<1){
				if(a[Ai-1]<b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_diag[p*33+i]=Ai-1;
					B_diag[p*33+i]=Bi+1;
					break;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi+1;
					break;
				}
			}

			if(a[Ai]>b[Bi-1]){
				if(a[Ai-1]<b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_top = Ai-1;
					B_top = Bi+1;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi+1;
					break;
				}
			}
			else if(a[Ai]<b[Bi-1]){
				A_bottom = Ai+1;
			}
			else if(a[Ai]==b[Bi-1]){
				A_diag[p*33+i]=Ai+1;
				B_diag[p*33+i]=Bi;
				break;
			}
		}

//		__syncthreads();
//merge
		index_t lowA  = A_diag[p*33+i];
		index_t lowB  = B_diag[p*33+i];
		index_t highA = A_diag[p*33+i+1];
		index_t highB = B_diag[p*33+i+1];
		vertex_t x,y;
		while(lowA<highA && lowB<highB){
//			iter++;//
			x=a[lowA];
			y=b[lowB];
			if(x<y){
				lowA++;
			}
			else if(x>y){
				lowB++;
			}
			else if(x==y){
				lowA++;
				lowB++;
				mycount++;
			}
		}
		tid += blockDim.x * gridDim.x/32;
//		__syncthreads();
	}
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<max_thd; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
//	iter_count[blockDim.x*blockIdx.x+threadIdx.x]=iter;
}


__global__ void block_merge_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
//	index_t*	iter_count,
	index_t*	count
)
{
//	index_t iter=0;
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/256 + Ns;
	int i = threadIdx.x;
	index_t mycount=0;
	__shared__ index_t local[max_thd];
	__shared__ index_t A_diag[257];
	__shared__ index_t B_diag[257];	
	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];
		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		if(i==0){
			A_diag[256]=m;
			B_diag[256]=n;
		}
		index_t index = (m+n)/256*i;
		index_t A_top, A_bottom, B_top, Ai, Bi;
		if(index>m){
			A_top = m;
			B_top = index-m;
		}
		else if(index<=m){
			A_top = index;
			B_top = 0;
		}
		if(index>n){
			A_bottom = index-n;
		}
		else if(index<=n){
			A_bottom = 0;
		}
//partition
		while(1){
			index_t offset=(A_top-A_bottom)/2;
			if(A_top==A_bottom){
				A_diag[i]=A_top;
				B_diag[i]=B_top;
				break;
			}

			Ai = A_top - offset;
			Bi = B_top + offset;
			if(offset<1){
				if(a[Ai-1]<b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_diag[i]=Ai-1;
					B_diag[i]=Bi+1;
					break;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi+1;
					break;
				}
			}

			if(a[Ai]>b[Bi-1]){
				if(a[Ai-1]<b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_top = Ai-1;
					B_top = Bi+1;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi+1;
					break;
				}
			}
			else if(a[Ai]<b[Bi-1]){
				A_bottom = Ai+1;
			}
			else if(a[Ai]==b[Bi-1]){
				A_diag[i]=Ai+1;
				B_diag[i]=Bi;
				break;
			}
		}

		__syncthreads();
//merge
		index_t lowA  = A_diag[i];
		index_t lowB  = B_diag[i];
		index_t highA = A_diag[i+1];
		index_t highB = B_diag[i+1];
		vertex_t x,y;
		while(lowA<highA && lowB<highB){
			x=a[lowA];
			y=b[lowB];
//			iter++;//
			if(x<y){
				lowA++;
			}
			else if(x>y){
				lowB++;
			}
			else if(x==y){
				lowA++;
				lowB++;
				mycount++;
			}
		}
		tid += blockDim.x * gridDim.x/256;
		__syncthreads();
	}
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<max_thd; i++){
			val+= local[i];
		}
		count[blockIdx.x] +=val;
	}
//	iter_count[blockDim.x*blockIdx.x+threadIdx.x]=iter;
}
__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
//	index_t*	degree,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ block_thd;
	int i = threadIdx.x% block_thd;
	index_t mycount=0;
//	__shared__ int cache[256];
//	__shared__ int offset[256];
	__shared__ index_t local[block_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];
		
		index_t temp;
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/block_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = block_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + block_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
//			bot = bot*k;
//			top = top*k;
			bot = bot*m/block_thd;
			top = top*m/block_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += block_thd;
		
		}
		tid += gridDim.x*blockDim.x/ block_thd;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
//	index_t*	degree,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[warp_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
//			bot = bot*k;
//			top = top*k;
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();

}

//----------------------------------------------------------------------------------------

__global__ void classify_kernel	//step 1: classify the edge list into different arrays
(	vertex_t* adj_list,
	vertex_t* head_list,
	index_t* begin,
	index_t  N,		//inputs
	index_t* small_num,
	index_t* mid_num,
	index_t* large_num
	//outputs: small/large head, adjacent, and number by thread
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space
	index_t small_offset=0;
	index_t mid_offset=0;
	index_t large_offset=0;
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
//			else{
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				large_offset++;
			}
		}
	}
	small_num[tid] = small_offset;
	mid_num[tid]   = mid_offset;
	large_num[tid] = large_offset;

}

__global__ void prefix_kernel_1	//this prefix scan function could be easier for data size is always 256*256
(	
 	index_t*	data,
	index_t*	block_offset
)
{
		
	//step 1: each block do prefix sum inside
	int tid = threadIdx.x +blockIdx.x*blockDim.x;

	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = data[tid];
	__syncthreads();

	index_t val=0;
	for(int i=0; i<=threadIdx.x; i++){
		val += temp_in[i];
	}


	__syncthreads();
	
	if(threadIdx.x==255){
		block_offset[blockIdx.x] = val;
		
	}
	data[tid] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_2	
(	
	index_t*	block_offset
)
{
	//step 2: collect each block's offset and do prefix for this set
	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = block_offset[threadIdx.x];
	__syncthreads();
	index_t val=0;
	for(int i=0; i<threadIdx.x; i++){
		val += temp_in[i];
	}
//		val = temp_in[threadIdx.x];
	block_offset[threadIdx.x] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_3	
(	
	index_t*	data,
	index_t*	block_offset
)
{
	//step 3: update by adding block offset
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	index_t val = data[tid];
	index_t offset = block_offset[blockIdx.x];
	val += offset;

	data[tid] = val;
	__syncthreads();
}

__global__ void collect_kernel
(	vertex_t* 	adj_list,
	vertex_t* 	head_list,
	index_t* 	begin,
	index_t	N,
	index_t* 	small_num,
	index_t* 	mid_num,
	index_t* 	large_num,
	index_t 	N1,
	index_t	N2,
	vertex_t*	dest_head,
	vertex_t*	dest_adj
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space


	index_t thd_base_small = 0;
	index_t thd_base_mid   = N1;
	index_t thd_base_large = N1+N2;
	if(tid!=0){
		thd_base_small = small_num[tid-1];
		thd_base_mid   = N1 + mid_num[tid-1];
		thd_base_large = N1 + N2 + large_num[tid-1];
	}
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	index_t small_offset = thd_base_small;
	index_t mid_offset   = thd_base_mid;
	index_t large_offset = thd_base_large;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				dest_head[small_offset] = head;
				dest_adj [small_offset] = adj;
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
//			else{
				dest_head[mid_offset] = head;
				dest_adj [mid_offset] = adj;
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				dest_head[large_offset] = head;
				dest_adj [large_offset] = adj;
				large_offset++;
			}
		}
	}
}


__global__ void reduce_kernel2(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void* part_scan(void * data){

	index_t thd_count=0;	
	int GPU_id = *(int*)data;
	int i = GPU_id;
//	cout<<"GPU id = "<<GPU_id<<"\n";
	hipSetDevice(GPU_id);
	H_ERR(hipDeviceSynchronize() );

	vertex_t*	gpu_adj;
	vertex_t*	gpu_head;
	index_t*	gpu_begin;
	index_t*	gpu_count;

	index_t partEdgeCount = mygraph->partEdgeCount[i];
	vertex_t vert_count = mygraph->vert_count;
	vertex_t* partAdj = mygraph->partAdj[i];
	vertex_t* partHead= mygraph->partHead[i];
	index_t* partBegin  = mygraph->partBegin[i];
	index_t* count    = mygraph->count;

	H_ERR(hipMalloc(&gpu_adj, partEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&gpu_head, partEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&gpu_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&gpu_count,    max_block*sizeof(index_t)) );

		
	
	index_t* block_offset;
	H_ERR(hipMalloc(&block_offset, max_block*sizeof(index_t)) );
	
	H_ERR(hipMemcpy(gpu_adj,    partAdj, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(gpu_head,   partHead, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(gpu_begin,  partBegin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );

	double time1=wtime();
	for(int j=0; j<PART_NUM; j++){	
		index_t totalEdgeCount = mygraph->partEdgeCount[j];
		vertex_t* 	head = mygraph->partHead[j];
		vertex_t* 	adj  = mygraph->partAdj[j];
		vertex_t*	classified_head;
		vertex_t*	classified_adj;
		
		index_t*	small_num;
		index_t*	mid_num;
		index_t*	large_num;

		vertex_t*	src_head;
		vertex_t*	src_adj;
		
		H_ERR(hipMalloc(&small_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&mid_num,   max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&large_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&src_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&src_adj,  totalEdgeCount*sizeof(vertex_t)) );
		
		H_ERR(hipMemcpy(src_adj,    adj, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(src_head,   head, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		
		H_ERR(hipMalloc(&classified_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&classified_adj,  totalEdgeCount*sizeof(vertex_t)) );
		//

		H_ERR(hipDeviceSynchronize() );

		
		classify_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					gpu_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num
					);
		H_ERR(hipDeviceSynchronize() );

		//test for prefix sum

		prefix_kernel_1 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

		prefix_kernel_1 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		
		prefix_kernel_1 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

	index_t N1,N2,N3;	
		H_ERR(hipMemcpy(&N1 ,  &small_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N2 , &mid_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N3 ,  &large_num[65535]   , sizeof(index_t), hipMemcpyDeviceToHost) );

		H_ERR(hipDeviceSynchronize() );
	//	cout<<"N1 = "<<N1<<"\n";
	//	cout<<"N2 = "<<N2<<"\n";
	//	cout<<"N3 = "<<N3<<"\n";
		
		collect_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					gpu_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num,
					N1,
					N2,
					classified_head,
					classified_adj
					);
		H_ERR(hipDeviceSynchronize() );


	//double time2=wtime();


		warp_merge_kernel<<<max_block,warp_thd>>>
		(	classified_head,
			classified_adj,
			gpu_adj,
//			dev_degree,
			gpu_begin,
			0,
			N1,
			gpu_count
		);
		H_ERR(hipDeviceSynchronize() );

	//double time3 = wtime();
		block_merge_kernel<<<max_block,max_thd>>>
		(	classified_head,
			classified_adj,
			gpu_adj,
			gpu_begin,
			N1,
			N1+N2,//totalEdgeCount,
			gpu_count
		);
		H_ERR(hipDeviceSynchronize() );
		
		reduce_kernel2 <<<1,1>>>(gpu_count);
		H_ERR(hipDeviceSynchronize() );
		
		H_ERR(hipMemcpy(&count[i], gpu_count, sizeof(index_t), hipMemcpyDeviceToHost));
		thd_count += count[i];
		
		H_ERR(hipFree(small_num) );
		H_ERR(hipFree(large_num) );
		H_ERR(hipFree(classified_head) );
		H_ERR(hipFree(classified_adj) );
		H_ERR(hipFree(src_head) );
		H_ERR(hipFree(src_adj) );
		cout<<"GPU "<<i<<" part "<<j<<"\n";
	}
	double time4 = wtime();

	count[i] = thd_count;
	cout<<"gpu binary count="<<count[i]<<"\n";
	cout<<"gpu time = "<<time4-time1<<endl;
	H_ERR(hipFree(gpu_adj) );
	H_ERR(hipFree(gpu_head) );
	H_ERR(hipFree(gpu_begin) );
	
	H_ERR(hipFree(block_offset) );
	H_ERR(hipFree(gpu_count) );
	return NULL;	
}



