#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 
#define thread_limit 256 
#define block_limit 1024 

#define GPU_COWORKER 1 
graph * mygraph;
long	total_count;

__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,

	long int*	counter_1,
	long int*	counter_2,

	index_t*	count
)
{
	int p = threadIdx.x/32;
	long counter1=0;
	long counter2=0;
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];



		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
//		local[i]=a[i*m/max_thd];	
		__syncthreads();

	counter1 += 8;
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
	counter1++;
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
/*
			while(top>bot+1){
				
		__syncthreads();
				warp_path[3*p]=0;
				warp_path[3*p+1]=0;
				warp_path[3*p+2]=0;
		__syncthreads();

				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + max_thd;
					warp_path[3*p]=1;
				}
				if(X<Y){
					top = r;
					warp_path[3*p+1]=1;
				}
				if(X>Y){
					bot = r;
					warp_path[3*p+2]=1;
				}

				int k=0;
				if(warp_path[3*p]!=0){
					k++;
				}
				if(warp_path[3*p+1]!=0){
					k++;
				}
				if(warp_path[3*p+2]!=0){
					k++;
				}
		counter2 +=k;

			}
*/
			//phase 2
//			bot = bot*m/max_thd;
//			top = top*m/max_thd -1;
			bot = 0;
			top = m-1;
			while(top>=bot){
		

				r = (top+bot)/2;
				Y = a[r];
	counter1++;
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
				
			
			}
			j += max_thd;
		
		}
		tid += GPU_COWORKER * gridDim.x*blockDim.x/ max_thd;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
	counter_1[blockDim.x*blockIdx.x+threadIdx.x]+=counter1;
	counter_2[blockDim.x*blockIdx.x+threadIdx.x]+=counter2;
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,

	long int*	counter_1,
	long int*	counter_2,

	index_t*	count
)
{
	long counter1=0;
	long counter2=0;
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
//		local[p*32+i]=a[i*m/32];	
	counter1+=8;
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
	counter1++;
			vertex_t Y;

			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
/*
			while(top>bot+1){
		__syncthreads();
				warp_path[3*p]=0;
				warp_path[3*p+1]=0;
				warp_path[3*p+2]=0;
		__syncthreads();
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
					warp_path[3*p]=1;
				}
				if(X<Y){
					top = r;
					warp_path[3*p+1]=1;
				}
				if(X>Y){
					bot = r;
					warp_path[3*p+2]=1;
				}
				int k=0;
				if(warp_path[3*p]!=0){
					k++;
				}
				if(warp_path[3*p+1]!=0){
					k++;
				}
				if(warp_path[3*p+2]!=0){
					k++;
				}
		counter2 +=k;
			}
*/
			//phase 2
			bot = 0;
			top = m -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
	counter1++;
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += GPU_COWORKER* blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();
	counter_1[blockDim.x*blockIdx.x+threadIdx.x]=counter1;
	counter_2[blockDim.x*blockIdx.x+threadIdx.x]=counter2;

}

//----------------------------------------------------------------------------------------

__global__ void reduce_kernel_count(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block*max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

__global__ void reduce_kernel(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void* part_scan(void * data){

	index_t thd_count=0;	
	int GPU_id = *(int*)data;
	int i = GPU_id;
//	cout<<"GPU id = "<<GPU_id<<"\n";
	hipSetDevice(GPU_id);
	H_ERR(hipDeviceSynchronize() );

	vertex_t*	dev_adj;
	index_t*	dev_begin;
	index_t*	dev_count;

	index_t partEdgeCount = mygraph->partEdgeCount[i];
	vertex_t vert_count = mygraph->vert_count;
	vertex_t* partAdj = mygraph->partAdj[i];
	vertex_t* partHead= mygraph->partHead[i];
	index_t* partBegin  = mygraph->partBegin[i];
	index_t* count    = mygraph->count;

	H_ERR(hipMalloc(&dev_adj, partEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

		
	
	index_t* block_offset;
	H_ERR(hipMalloc(&block_offset, max_block*sizeof(index_t)) );
	
	H_ERR(hipMemcpy(dev_adj,    partAdj, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  partBegin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );
	
long 	counter_1_cpu=0;
//long 	counter_2_cpu=0;
long int tmp_counter1,tmp_counter2;
long int*       counter_1;//counter for memory read
long int*       counter_2;//counter for divergence
H_ERR(hipMalloc(&counter_1,    max_thd*max_block*sizeof(long int)) );
H_ERR(hipMalloc(&counter_2,    max_thd*max_block*sizeof(long int)) );

	double time2=wtime();
	for(int j=0; j<PART_NUM; j++){	
		index_t totalEdgeCount = mygraph->partEdgeCount[j];
		vertex_t* 	head = mygraph->partHead[j];
		vertex_t* 	adj  = mygraph->partAdj[j];

		vertex_t*	src_head;
		vertex_t*	src_adj;
		
		H_ERR(hipMalloc(&src_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&src_adj,  totalEdgeCount*sizeof(vertex_t)) );
		
		H_ERR(hipMemcpy(src_adj,    adj, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(src_head,   head, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		
		//

	double time1=wtime();
		H_ERR(hipDeviceSynchronize() );

		
		warp_binary_kernel<<<max_block,max_thd>>>
		(	src_head,
			src_adj,
			dev_adj,
			dev_begin,
			0,
			totalEdgeCount,
			
			counter_1,
			counter_2,

			dev_count
		);
		H_ERR(hipDeviceSynchronize() );
		
		reduce_kernel <<<1,1>>>(dev_count);
		H_ERR(hipDeviceSynchronize() );
		
		H_ERR(hipMemcpy(&count[i], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
		thd_count += count[i];
		
		
		reduce_kernel_count <<<1,1>>>(counter_1);
		H_ERR(hipDeviceSynchronize() );
//		reduce_kernel_count <<<1,1>>>(counter_2);
//		H_ERR(hipDeviceSynchronize() );
		//long int tmp_counter1,tmp_counter2;
		H_ERR(hipMemcpy(&tmp_counter1, counter_1, sizeof(long), hipMemcpyDeviceToHost));
//		H_ERR(hipMemcpy(&tmp_counter2, counter_2, sizeof(long), hipMemcpyDeviceToHost));
		counter_1_cpu += tmp_counter1;
//		counter_2_cpu += tmp_counter2;
	
	
		H_ERR(hipFree(src_head) );
		H_ERR(hipFree(src_adj) );
//		H_ERR(hipFree(src_begin) );
//		cout<<"GPU "<<i<<" part "<<j<<"\n";
	}
	
	double time4 = wtime();
	count[i] = thd_count;
//	cout<<"gpu "<<i<<" binary count="<<count[i]<<"\n";
//	cout<<"time = "<<time4-time2<<" seconds"<<endl;

//	cout<<"counter for mem_read   = "<<counter_1_cpu<<endl;
//	cout<<"counter for divergence = "<<counter_2_cpu<<endl;
	total_count += counter_1_cpu;

	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(block_offset) );
	H_ERR(hipFree(dev_count) );
	return NULL;	
}


