#include "hip/hip_runtime.h"
//Graph format: Json based format
//Storage format: 
//struct{
//		int: src_ver
//		Arr: [ver_0|ver_1|ver_2|...]
//		Int: num_conn_ver
//	}
/* main.cu */
#include "graph.h"
#include <sstream>
#include <iostream>
#include <fstream>
#include <pthread.h>
#define N 256*256
using namespace std;



int main(int args, char *argv[]) {
//	pthread_t thd1;
	std::cout<<"Input format: ./exe graph-file-name"
						<<" (json formated file)\n";

	if(args != 2) return -1;
	string json_file 	= argv[1];
	graph *graph_d 
		= new graph	(json_file); 
	
	cout<<"GPU  NUMBER = "<<GPU_NUM<<endl;
	cout<<"PART NUMBER = "<<PART_NUM<<endl;

	mygraph=graph_d;
	cout<<"rank by degree\n";

	double total_t=0;
int r=2;
for(int n=0; n<r; n++){
	//initiation, clean the footprint of last executionfor(int i=0; i<GPU_NUM+1; i++){
	for(int i=0; i<DEV_NUM; i++){
		graph_d->ds_complete[i]=0;
		graph_d->ds_help[i]=0;
	}

#pragma omp parallel for
	for(int i=0; i<PART_NUM * graph_d->ChunkNum; i++){
		graph_d->ds_status[i]=0;
	}

	double t0=wtime();
//	index_t total = 0;

//int tid;
omp_set_nested(1);
{
#pragma omp parallel for num_threads(DEV_NUM) schedule(static)
	for(int i=0; i<DEV_NUM; i++){
		if(i<GPU_NUM){
			graph_d->gpuProc(i);
		}
	}
}

	
//	for(int i=0; i<GPU_NUM+1; i++){
//		total+= graph_d->count[i];
//	}
	graph_d->reduceResult();
	double t1=wtime();
	cout<<"total count "<<graph_d->count[0]<<"\n";
	cout<<"total time  "<<t1-t0<<" seconds\n";
	total_t += t1-t0;
}
cout<<"merge average time of 5 round = "<<total_t/r<<endl;
	return 0;
}
