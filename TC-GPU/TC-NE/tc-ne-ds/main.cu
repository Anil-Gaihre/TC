#include "hip/hip_runtime.h"
//Graph format: Json based format
//Storage format: 
//struct{
//		int: src_ver
//		Arr: [ver_0|ver_1|ver_2|...]
//		Int: num_conn_ver
//	}
/* main.cu */
#include "graph.h"
#include <sstream>
#include <iostream>
#include <fstream>
#include <pthread.h>
#define N 256*256
using namespace std;



int main(int args, char *argv[]) {
//	pthread_t thd1;
	std::cout<<"Input format: ./exe graph-file-name"
						<<" (json formated file)\n";

	if(args != 2) return -1;
	string json_file 	= argv[1];
	graph *graph_d 
		= new graph	(json_file); 
	mygraph=graph_d;
	cout<<"no edge reduction\n";
// test sort part!
//	graph_d->sort();
//	graph_d->reduce();
//	cout<<"rank-by-degree"<<endl;
//	graph_d->rank_by_degree();
//	graph_d->reverse_rank_by_degree();
	graph_d->partition();
//	graph_d->validation();
/*
	graph_d->triangle_count();
	graph_d->validation();

	int err=0;
	int count1=graph_d->count[0];
	int count2=0;
	for(int i=0; i<graph_d->vert_count; i++){
		count2+= graph_d->valid[i];
	}
	err = count1-count2;
	printf("count1 = %d, count2 = %d\n",count1,count2);
	printf("err number = %d\n",err);
*/
	double total_t=0;
for(int n=0; n<5; n++){
	for(int i=0; i<GPU_NUM*GPU_NUM; i++){
		graph_d->ds_status[i]=0;
		graph_d->ds_count[i]=0;
	}
	for(int i=0; i<GPU_NUM; i++){
		graph_d->ds_complete[i]=0;
		graph_d->ds_help[i]=0;
	}
	for(int i=0;i<GPU_NUM;i++){
		graph_d->count[i]=0;
	}
// test scan part!'
	pthread_t *thd = new pthread_t[GPU_NUM];
	double t0=wtime();
	int id[GPU_NUM];
	for(int i=0; i<GPU_NUM;i++){
//		cout<<"loop "<<i<<"\n";
		id[i] = i;
		pthread_create(&thd[i],NULL,part_scan,&id[i]);
//		part_scan(&id[i]);
	}

	for(int i=0; i<GPU_NUM;i++){
		pthread_join(thd[i],NULL);
	}

	long int total=0;
	for(int i=0;i<GPU_NUM;i++){
		total+= graph_d->count[i];
	}
	double t1=wtime();
//	cout<<"total count "<<total<<"\n";
	//test
	total=0;
	for(int i=0;i<GPU_NUM*GPU_NUM;i++){
		total+= graph_d->ds_count[i];
	}
	cout<<"total count "<<total/6<<"\n";

	cout<<"total time  "<<t1-t0<<" seconds\n";
total_t += t1-t0;
}
cout<<"average time of 5 round = "<<total_t/5<<endl;
//	cout<<"total count "<<total<<"\n";

//	cout<<"total time  "<<t1-t0<<" seconds\n";

	return 0;
}
