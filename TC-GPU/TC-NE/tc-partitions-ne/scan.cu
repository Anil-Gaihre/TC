#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 
#define thread_limit 256 
#define block_limit 1024 

#define GPU_COWORKER 1 
graph * mygraph;

__global__ void warp_merge_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/32;
	int i = threadIdx.x%32;
	int p = threadIdx.x/32;
	long int mycount=0;
	__shared__ index_t local[max_thd];
	__shared__ vertex_t A_diag[33*8];
	__shared__ vertex_t B_diag[33*8];	
	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
		if(i==0){
			A_diag[p*33+32]=m;
			B_diag[p*33+32]=n;
		}
		index_t index = (m+n)/32*i;
		vertex_t A_top, A_bottom, B_top, Ai, Bi;
		if(index>m){
			A_top = m;
			B_top = index-m;
		}
		else if(index<=m){
			A_top = index;
			B_top = 0;
		}
		if(index>n){
			A_bottom = index-n;
		}
		else if(index<=n){
			A_bottom = 0;
		}

		while(1){
			int offset=(A_top-A_bottom)/2;
			if(A_top==A_bottom){
				A_diag[p*33+i]=A_top;
				B_diag[p*33+i]=B_top;
				break;
			}

			Ai = A_top - offset;
			Bi = B_top + offset;
			if(offset<1){
				if(a[Ai-1]<b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_diag[p*33+i]=Ai-1;
					B_diag[p*33+i]=Bi+1;
					break;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi+1;
					break;
				}
			}

			if(a[Ai]>b[Bi-1]){
				if(a[Ai-1]<b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_top = Ai-1;
					B_top = Bi+1;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi+1;
					break;
				}
			}
			else if(a[Ai]<b[Bi-1]){
				A_bottom = Ai+1;
			}
			else if(a[Ai]==b[Bi-1]){
				A_diag[p*33+i]=Ai+1;
				B_diag[p*33+i]=Bi;
				break;
			}
		}

//		__syncthreads();

		vertex_t lowA  = A_diag[p*33+i];
		vertex_t lowB  = B_diag[p*33+i];
		vertex_t highA = A_diag[p*33+i+1];
		vertex_t highB = B_diag[p*33+i+1];
		vertex_t x,y;
		while(lowA<highA && lowB<highB){
			x=a[lowA];
			y=b[lowB];
			if(x<y){
				lowA++;
			}
			else if(x>y){
				lowB++;
			}
			else if(x==y){
				lowA++;
				lowB++;
				mycount++;
			}
		}
		tid += blockDim.x * gridDim.x/32;
//		__syncthreads();
	}
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();
}


__global__ void block_merge_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/256;
	int i = threadIdx.x;
	index_t mycount=0;
	__shared__ index_t local[max_thd];
	__shared__ vertex_t A_diag[257];
	__shared__ vertex_t B_diag[257];	
	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		if(i==0){
			A_diag[256]=m;
			B_diag[256]=n;
		}
		index_t index = (m+n)/256*i;
		vertex_t A_top, A_bottom, B_top, Ai, Bi;
		if(index>m){
			A_top = m;
			B_top = index-m;
		}
		else if(index<=m){
			A_top = index;
			B_top = 0;
		}
		if(index>n){
			A_bottom = index-n;
		}
		else if(index<=n){
			A_bottom = 0;
		}

		while(1){
			int offset=(A_top-A_bottom)/2;
			if(A_top==A_bottom){
				A_diag[i]=A_top;
				B_diag[i]=B_top;
				break;
			}

			Ai = A_top - offset;
			Bi = B_top + offset;
			if(offset<1){
				if(a[Ai-1]<b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_diag[i]=Ai-1;
					B_diag[i]=Bi+1;
					break;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi+1;
					break;
				}
			}

			if(a[Ai]>b[Bi-1]){
				if(a[Ai-1]<b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_top = Ai-1;
					B_top = Bi+1;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi+1;
					break;
				}
			}
			else if(a[Ai]<b[Bi-1]){
				A_bottom = Ai+1;
			}
			else if(a[Ai]==b[Bi-1]){
				A_diag[i]=Ai+1;
				B_diag[i]=Bi;
				break;
			}
		}

		__syncthreads();

		vertex_t lowA  = A_diag[i];
		vertex_t lowB  = B_diag[i];
		vertex_t highA = A_diag[i+1];
		vertex_t highB = B_diag[i+1];
		vertex_t x,y;
		while(lowA<highA && lowB<highB){
			x=a[lowA];
			y=b[lowB];
			if(x<y){
				lowA++;
			}
			else if(x>y){
				lowB++;
			}
			else if(x==y){
				lowA++;
				lowB++;
				mycount++;
			}
}
		tid += blockDim.x * gridDim.x/256;
		__syncthreads();
	}
	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
}
__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];


		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/max_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + max_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/max_thd;
			top = top*m/max_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += max_thd;
		
		}
		tid += GPU_COWORKER * gridDim.x*blockDim.x/ max_thd;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += GPU_COWORKER* blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();

}

//----------------------------------------------------------------------------------------

__global__ void classify_kernel	//step 1: classify the edge list into different arrays
(	vertex_t* adj_list,
	vertex_t* head_list,
	index_t* begin,
	index_t  N,		//inputs
	index_t* small_num,
	index_t* mid_num,
	index_t* large_num
	//outputs: small/large head, adjacent, and number by thread
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space
	index_t small_offset=0;
	index_t mid_offset=0;
	index_t large_offset=0;
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
//			else{
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				large_offset++;
			}
		}
	}
	small_num[tid] = small_offset;
	mid_num[tid]   = mid_offset;
	large_num[tid] = large_offset;

}

__global__ void prefix_kernel_1	//this prefix scan function could be easier for data size is always 256*256
(	
 	index_t*	data,
	index_t*	block_offset
)
{
		
	//step 1: each block do prefix sum inside
	int tid = threadIdx.x +blockIdx.x*blockDim.x;

	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = data[tid];
	__syncthreads();

	index_t val=0;
	for(int i=0; i<=threadIdx.x; i++){
		val += temp_in[i];
	}


	__syncthreads();
	
	if(threadIdx.x==255){
		block_offset[blockIdx.x] = val;
		
	}
	data[tid] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_2	
(	
	index_t*	block_offset
)
{
	//step 2: collect each block's offset and do prefix for this set
	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = block_offset[threadIdx.x];
	__syncthreads();
	index_t val=0;
	for(int i=0; i<threadIdx.x; i++){
		val += temp_in[i];
	}
//		val = temp_in[threadIdx.x];
	block_offset[threadIdx.x] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_3	
(	
	index_t*	data,
	index_t*	block_offset
)
{
	//step 3: update by adding block offset
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	index_t val = data[tid];
	index_t offset = block_offset[blockIdx.x];
	val += offset;

	data[tid] = val;
	__syncthreads();
}

__global__ void collect_kernel
(	vertex_t* 	adj_list,
	vertex_t* 	head_list,
	index_t* 	begin,
	index_t	N,
	index_t* 	small_num,
	index_t* 	mid_num,
	index_t* 	large_num,
	index_t 	N1,
	index_t	N2,
	vertex_t*	dest_head,
	vertex_t*	dest_adj
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space


	index_t thd_base_small = 0;
	index_t thd_base_mid   = N1;
	index_t thd_base_large = N1+N2;
	if(tid!=0){
		thd_base_small = small_num[tid-1];
		thd_base_mid   = N1 + mid_num[tid-1];
		thd_base_large = N1 + N2 + large_num[tid-1];
	}
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	index_t small_offset = thd_base_small;
	index_t mid_offset   = thd_base_mid;
	index_t large_offset = thd_base_large;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				dest_head[small_offset] = head;
				dest_adj [small_offset] = adj;
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
//			else{
				dest_head[mid_offset] = head;
				dest_adj [mid_offset] = adj;
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				dest_head[large_offset] = head;
				dest_adj [large_offset] = adj;
				large_offset++;
			}
		}
	}
}


__global__ void reduce_kernel2(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void* part_scan(void * data){

	index_t thd_count=0;	
	int GPU_id = *(int*)data;
	int i = GPU_id;
//	cout<<"GPU id = "<<GPU_id<<"\n";
//	hipSetDevice(GPU_id);
	hipSetDevice(1);

	H_ERR(hipDeviceSynchronize() );

	vertex_t*	dev_adj;
//	vertex_t*	dev_head;
	index_t*	dev_begin;
	index_t*	dev_count;

	index_t partEdgeCount = mygraph->partEdgeCount[i];
	vertex_t vert_count = mygraph->vert_count;
	vertex_t* partAdj = mygraph->partAdj[i];
//	vertex_t* partHead= mygraph->partHead[i];
//	index_t* partDegree = mygraph->partDegree[i];
	index_t* partBegin  = mygraph->partBegin[i];
	index_t* count    = mygraph->count;

	H_ERR(hipMalloc(&dev_adj, partEdgeCount*sizeof(vertex_t)) );
//	H_ERR(hipMalloc(&dev_head, partEdgeCount*sizeof(vertex_t)) );
//	H_ERR(hipMalloc(&dev_degree, vert_count*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

		
	
	index_t* block_offset;
	H_ERR(hipMalloc(&block_offset, max_block*sizeof(index_t)) );
	
	H_ERR(hipMemcpy(dev_adj,    partAdj, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
//	H_ERR(hipMemcpy(dev_head,   partHead, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
//	H_ERR(hipMemcpy(dev_degree, partDegree, vert_count*sizeof(index_t),  hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  partBegin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );

	double time2=wtime();
	for(int j=0; j<PART_NUM; j++){	
		index_t totalEdgeCount = mygraph->partEdgeCount[j];
		vertex_t* 	head = mygraph->partHead[j];
		vertex_t* 	adj  = mygraph->partAdj[j];
//		index_t* 	degree = mygraph->partDegree[j];
		vertex_t*	classified_head;
		vertex_t*	classified_adj;
		
		index_t*	small_num;
		index_t*	mid_num;
		index_t*	large_num;

		vertex_t*	src_head;
		vertex_t*	src_adj;
//		index_t*	src_degree;
		
		H_ERR(hipMalloc(&small_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&mid_num,   max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&large_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&src_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&src_adj,  totalEdgeCount*sizeof(vertex_t)) );
//		H_ERR(hipMalloc(&src_degree,  vert_count*sizeof(index_t)) );
		
		H_ERR(hipMemcpy(src_adj,    adj, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(src_head,   head, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
//		H_ERR(hipMemcpy(src_degree, degree, vert_count*sizeof(index_t), hipMemcpyHostToDevice) );
		
		H_ERR(hipMalloc(&classified_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&classified_adj,  totalEdgeCount*sizeof(vertex_t)) );
		//

	double time1=wtime();
		H_ERR(hipDeviceSynchronize() );

		
		classify_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					dev_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num
					);
		H_ERR(hipDeviceSynchronize() );

		//test for prefix sum

		prefix_kernel_1 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

		prefix_kernel_1 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		
		prefix_kernel_1 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

	index_t N1,N2,N3;	
		H_ERR(hipMemcpy(&N1 ,  &small_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N2 , &mid_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N3 ,  &large_num[65535]   , sizeof(index_t), hipMemcpyDeviceToHost) );

		H_ERR(hipDeviceSynchronize() );
	//	cout<<"N1 = "<<N1<<"\n";
	//	cout<<"N2 = "<<N2<<"\n";
	//	cout<<"N3 = "<<N3<<"\n";
		
		collect_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					dev_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num,
					N1,
					N2,
					classified_head,
					classified_adj
					);
		H_ERR(hipDeviceSynchronize() );




		warp_merge_kernel<<<max_block,max_thd>>>
		(	classified_head,
			classified_adj,
			dev_adj,
//			dev_degree,
			dev_begin,
			0,
			N1,
			dev_count
		);
		H_ERR(hipDeviceSynchronize() );

		block_merge_kernel<<<max_block,max_thd>>>
		(	classified_head,
			classified_adj,
			//dev_head,
			//dev_adj,

			dev_adj,
//			dev_degree,
			dev_begin,
			N1,
			N1+N2,
	//		0 + GPU_id*256,
	//		totalEdgeCount,
			dev_count
		);
		H_ERR(hipDeviceSynchronize() );
		
		reduce_kernel2 <<<1,1>>>(dev_count);
		H_ERR(hipDeviceSynchronize() );
		
		H_ERR(hipMemcpy(&count[i], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
		thd_count += count[i];
		
		H_ERR(hipFree(small_num) );
		H_ERR(hipFree(large_num) );
		H_ERR(hipFree(classified_head) );
		H_ERR(hipFree(classified_adj) );
		H_ERR(hipFree(src_head) );
		H_ERR(hipFree(src_adj) );
//		H_ERR(hipFree(src_begin) );
//		cout<<"GPU "<<i<<" part "<<j<<"\n";
	}
	
	double time4 = wtime();
	count[i] = thd_count;
//	cout<<"gpu "<<i<<" binary count="<<count[i]<<"\n";
//	cout<<"time = "<<time4-time2<<" seconds"<<endl;
	H_ERR(hipFree(dev_adj) );
//	H_ERR(hipFree(dev_head) );
//	H_ERR(hipFree(dev_degree) );
	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(block_offset) );
	H_ERR(hipFree(dev_count) );
	return NULL;	
}


