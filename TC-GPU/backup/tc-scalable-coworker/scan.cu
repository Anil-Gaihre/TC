#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 

graph * mygraph;

__global__ void warp_merge_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/32;
	int i = threadIdx.x%32;
	int p = threadIdx.x/32;
	long int mycount=0;
	__shared__ index_t local[max_thd];
	__shared__ vertex_t A_diag[33*8];
	__shared__ vertex_t B_diag[33*8];	
	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
		if(i==0){
			A_diag[p*33+32]=m;
			B_diag[p*33+32]=n;
		}
		index_t index = (m+n)/32*i;
		vertex_t A_top, A_bottom, B_top, Ai, Bi;
		if(index>m){
			A_top = m;
			B_top = index-m;
		}
		else if(index<=m){
			A_top = index;
			B_top = 0;
		}
		if(index>n){
			A_bottom = index-n;
		}
		else if(index<=n){
			A_bottom = 0;
		}

		while(1){
			int offset=(A_top-A_bottom)/2;
			if(A_top==A_bottom){
				A_diag[p*33+i]=A_top;
				B_diag[p*33+i]=B_top;
				break;
			}

			Ai = A_top - offset;
			Bi = B_top + offset;
			if(offset<1){
				if(a[Ai-1]<b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_diag[p*33+i]=Ai-1;
					B_diag[p*33+i]=Bi+1;
					break;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi+1;
					break;
				}
			}

			if(a[Ai]>b[Bi-1]){
				if(a[Ai-1]<b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_top = Ai-1;
					B_top = Bi+1;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi+1;
					break;
				}
			}
			else if(a[Ai]<b[Bi-1]){
				A_bottom = Ai+1;
			}
			else if(a[Ai]==b[Bi-1]){
				A_diag[p*33+i]=Ai+1;
				B_diag[p*33+i]=Bi;
				break;
			}
		}

//		__syncthreads();

		vertex_t lowA  = A_diag[p*33+i];
		vertex_t lowB  = B_diag[p*33+i];
		vertex_t highA = A_diag[p*33+i+1];
		vertex_t highB = B_diag[p*33+i+1];
		vertex_t x,y;
		while(lowA<highA && lowB<highB){
			x=a[lowA];
			y=b[lowB];
			if(x<y){
				lowA++;
			}
			else if(x>y){
				lowB++;
			}
			else if(x==y){
				lowA++;
				lowB++;
				mycount++;
			}
		}
//		tid += blockDim.x * gridDim.x/32;
		tid += GPU_PER_PART * gridDim.x*blockDim.x/32;
		
//		__syncthreads();
	}
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();
}


__global__ void block_merge_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/256;
	int i = threadIdx.x;
	index_t mycount=0;
	__shared__ index_t local[max_thd];
	__shared__ vertex_t A_diag[257];
	__shared__ vertex_t B_diag[257];	
	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		if(i==0){
			A_diag[256]=m;
			B_diag[256]=n;
		}
		index_t index = (m+n)/256*i;
		vertex_t A_top, A_bottom, B_top, Ai, Bi;
		if(index>m){
			A_top = m;
			B_top = index-m;
		}
		else if(index<=m){
			A_top = index;
			B_top = 0;
		}
		if(index>n){
			A_bottom = index-n;
		}
		else if(index<=n){
			A_bottom = 0;
		}

		while(1){
			int offset=(A_top-A_bottom)/2;
			if(A_top==A_bottom){
				A_diag[i]=A_top;
				B_diag[i]=B_top;
				break;
			}

			Ai = A_top - offset;
			Bi = B_top + offset;
			if(offset<1){
				if(a[Ai-1]<b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_diag[i]=Ai-1;
					B_diag[i]=Bi+1;
					break;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi+1;
					break;
				}
			}

			if(a[Ai]>b[Bi-1]){
				if(a[Ai-1]<b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_top = Ai-1;
					B_top = Bi+1;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[i]=Ai;
					B_diag[i]=Bi+1;
					break;
				}
			}
			else if(a[Ai]<b[Bi-1]){
				A_bottom = Ai+1;
			}
			else if(a[Ai]==b[Bi-1]){
				A_diag[i]=Ai+1;
				B_diag[i]=Bi;
				break;
			}
		}

		__syncthreads();

		vertex_t lowA  = A_diag[i];
		vertex_t lowB  = B_diag[i];
		vertex_t highA = A_diag[i+1];
		vertex_t highB = B_diag[i+1];
		vertex_t x,y;
		while(lowA<highA && lowB<highB){
			x=a[lowA];
			y=b[lowB];
			if(x<y){
				lowA++;
			}
			else if(x>y){
				lowB++;
			}
			else if(x==y){
				lowA++;
				lowB++;
				mycount++;
			}
}
//		tid += blockDim.x * gridDim.x/256;
		tid += GPU_PER_PART * gridDim.x*blockDim.x/256;
		__syncthreads();
	}
	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
}
__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];


		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/max_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + max_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/max_thd;
			top = top*m/max_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += max_thd;
		
		}
		tid += GPU_PER_PART * gridDim.x*blockDim.x/256;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
//		count[blockIdx.x]+=val;
		count[blockIdx.x]=val;
	}
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += GPU_PER_PART* blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();

}



__global__ void reduce_kernel2(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void part_scan(int i){

	index_t thd_count=0;	
	int P = i%PART_NUM;
	hipSetDevice(i);

	H_ERR(hipDeviceSynchronize() );

	vertex_t*	dev_adj;
//	vertex_t*	dev_head;
	index_t*	dev_begin;
	index_t*	dev_count;

	index_t partEdgeCount = mygraph->partEdgeCount[P];
	vertex_t vert_count = mygraph->vert_count;
	vertex_t* partAdj = mygraph->partAdj[P];
	index_t* partBegin  = mygraph->partBegin[P];
	index_t* count    = mygraph->count;

	H_ERR(hipMalloc(&dev_adj, partEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

	H_ERR(hipMemcpy(dev_adj,    partAdj, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  partBegin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );

	double time2=wtime();
	for(int j=0; j<PART_NUM; j++){	
		index_t totalEdgeCount = mygraph->partEdgeCount[j];
		vertex_t* 	head = mygraph->partHead[j];
		vertex_t* 	adj  = mygraph->partAdj[j];
		
		vertex_t*	src_head;
		vertex_t*	src_adj;
		
		H_ERR(hipMalloc(&src_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&src_adj,  totalEdgeCount*sizeof(vertex_t)) );
		
		H_ERR(hipMemcpy(src_adj,    adj, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(src_head,   head, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		

		H_ERR(hipDeviceSynchronize() );
		warp_binary_kernel<<<max_block,max_thd>>>
		(	src_head,
			src_adj,
			dev_adj,
			dev_begin,
			i/PART_NUM*256*256/32,
			totalEdgeCount,
			dev_count
		);
		H_ERR(hipDeviceSynchronize() );
		reduce_kernel2 <<<1,1>>>(dev_count);
		H_ERR(hipDeviceSynchronize() );
		
		H_ERR(hipMemcpy(&count[i], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
		thd_count += count[i];
		
		H_ERR(hipFree(src_head) );
		H_ERR(hipFree(src_adj) );
//		cout<<"GPU "<<i<<" part "<<j<<"\n";
	}
	
	double time4 = wtime();
	count[i] = thd_count;
//	cout<<"gpu "<<i<<" binary count="<<count[i]<<"\n";
//	cout<<"time = "<<time4-time2<<" seconds"<<endl;
	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(dev_count) );
}




