#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include <stdio.h>
#include "iostream"
#define max_thd 256 
#define max_block 256 

graph * mygraph;
__global__ void block_binary_kernel
(	//vertex_t*	head,
	//vertex_t*	adj,
	Edge*		workload,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
//		vertex_t A = head[tid];
//		vertex_t B = adj[tid];
		vertex_t A = workload[tid].A;
		vertex_t B = workload[tid].B;
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];


		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/max_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
//printf("find A %d B %d C %d\n",A,B,X);
					mycount++;
					bot = top + max_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/max_thd;
			top = top*m/max_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += max_thd;
		
		}
		tid += GPU_PER_PART * gridDim.x*blockDim.x/256;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
//		count[blockIdx.x]+=val;
		count[blockIdx.x]=val;
//		if(val!=0)
//			printf("+ %d\n",count[blockIdx.x]);
	}
}

__global__ void warp_binary_kernel
(	//vertex_t*	head,
	//vertex_t*	adj,
	Edge*		workload,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = workload[tid].A;
		vertex_t B = workload[tid].B;
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];
//if(i==0) printf("A %d B %d\n");
		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
//		tid += GPU_NUM* blockDim.x*gridDim.x/32;
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
//		count[blockIdx.x]=val;
		count[blockIdx.x]+=val;
	}
	__syncthreads();

}


__global__ void init_count(index_t* count)
{
	int tid = threadIdx.x;
	count[tid] = 0;
}

__global__ void reduce_kernel(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}


//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------



void graph::initDevice(int GPU_id,int Part_id){
//cuda memory copy of partAdj and partBegin
	hipSetDevice(GPU_id);

	int P=Part_id;
	H_ERR(hipDeviceSynchronize() );


	vertex_t*	dev_adj;		
	index_t*	dev_begin;	
	index_t*	dev_count;	
	Edge*		buffer0;	
	Edge*		buffer1;	

	index_t EdgeCount = partEdgeCount[P];
	vertex_t* Adj = partAdj[P];
	index_t* Begin  = partBegin[P];
	
	H_ERR(hipMalloc(&dev_adj, EdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

	H_ERR(hipMemcpy(dev_adj,    Adj, EdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  Begin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );
	
	H_ERR(hipMalloc(&buffer0,    BufferSize*sizeof(Edge)) );
	H_ERR(hipMalloc(&buffer1,    BufferSize*sizeof(Edge)) );
	
	gdata[GPU_id].adj	=	dev_adj;
	gdata[GPU_id].begin	=	dev_begin;
	gdata[GPU_id].count	=	dev_count;
	gdata[GPU_id].EdgeBuffer[0]=	buffer0;
	gdata[GPU_id].EdgeBuffer[1]=	buffer1;
	gdata[GPU_id].partition_id =	P;
	gdata[GPU_id].currentBuffer=	0;
	init_count <<<1,max_thd>>>(dev_count);

	ds_complete[GPU_id] = ChunkNum/(GPU_NUM+1)*(GPU_id);
	ds_last[GPU_id] = ChunkNum/(GPU_NUM+1)*(GPU_id+1);

}

void graph::DeviceCompute(int GPU_id, index_t Chunk_id){
	
	int P = gdata[GPU_id].partition_id;
	
	vertex_t*	dev_adj		=gdata[GPU_id].adj;
	index_t*	dev_begin	=gdata[GPU_id].begin;
	index_t*	dev_count	=gdata[GPU_id].count;
	Edge*		buffer		=gdata[GPU_id].EdgeBuffer[gdata[GPU_id].currentBuffer%2];
	gdata[GPU_id].currentBuffer	=1-gdata[GPU_id].currentBuffer;
	index_t currentBufferSize = BufferSize;
	if(Chunk_id==upperEdgeCount/BufferSize){
		currentBufferSize = upperEdgeCount % BufferSize;
	}
	init_count <<<1,max_thd>>>(dev_count);
	H_ERR(hipMemcpy(buffer, &OrientedEdge[Chunk_id*BufferSize], currentBufferSize*sizeof(Edge), hipMemcpyHostToDevice) );
	H_ERR(hipDeviceSynchronize() );
	warp_binary_kernel<<<max_block,max_thd>>>
	(	buffer,
		dev_adj,
		dev_begin,
		0,
//		GPU_id*256*256/32,
		currentBufferSize,
		dev_count
	);
	//write the result of this chunk back
	H_ERR(hipDeviceSynchronize() );
	index_t tempcount[max_block];
	index_t mycount=0;
	H_ERR(hipMemcpy(tempcount, dev_count, max_block*sizeof(index_t), hipMemcpyDeviceToHost));
	for(int i=0; i<max_block; i++) mycount += tempcount[i];
	ds_count[P * ChunkNum + Chunk_id] = mycount;
}

void graph::gpuReduce(int GPU_id){
	vertex_t*	dev_adj		=gdata[GPU_id].adj;
	index_t*	dev_begin	=gdata[GPU_id].begin;
	index_t*	dev_count	=gdata[GPU_id].count;
	Edge**		buffer		=gdata[GPU_id].EdgeBuffer;
//	H_ERR(hipDeviceSynchronize() );
//	reduce_kernel <<<1,max_thd>>>(dev_count);
//	H_ERR(hipMemcpy(&count[GPU_id], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
//		thd_count += count[i];
//	count[i] = thd_count;
	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_begin) );
	H_ERR(hipFree(dev_count) );
	H_ERR(hipFree(buffer[0]) );
	H_ERR(hipFree(buffer[1]) );
//	cout<<"GPU "<<GPU_id<<" finished"<<endl;
}

void graph::gpuProc(int GPU_id){
double t0 = wtime();
//	index_t total_count=0;
//	step 1: computing 
	for(int P=0; P<PART_NUM; P++){
		initDevice(GPU_id,P);
//		cout<<"GPU "<<GPU_id<<" last chunk "<<ds_last[P][GPU_id]<<endl;
		for(index_t i=GPU_id; i<ChunkNum; i+=DEV_NUM ){
			if(ds_status[P*ChunkNum + i]!=0){ 
				return;//break;
			}
			//finish with someone's help
			//
			ds_status[P*ChunkNum + i] = 1;
			ds_complete[GPU_id]++;
			DeviceCompute(GPU_id,i);
//cout<<"GPU "<<GPU_id<<" chunk "<<i<<endl;

		}
	}
//step 2: work stealing
//-------------
//	count[GPU_id] = total_count;
	gpuReduce(GPU_id);
double t1 = wtime();
cout<<"GPU "<<GPU_id<<" time = "<<t1-t0<<endl;
//cout<<"GPU complete = "<<ds_complete[GPU_id];
}
