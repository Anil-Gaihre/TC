#include "hip/hip_runtime.h"
//Graph format: Json based format
//Storage format: 
//struct{
//		int: src_ver
//		Arr: [ver_0|ver_1|ver_2|...]
//		Int: num_conn_ver
//	}
/* main.cu */
#include "graph.h"
#include <sstream>
#include <iostream>
#include <fstream>
#include <pthread.h>
#define N 256*256
using namespace std;



int main(int args, char *argv[]) {
//	pthread_t thd1;
	std::cout<<"Input format: ./exe graph-file-name"
						<<" (json formated file)\n";

	if(args != 2) return -1;
	string json_file 	= argv[1];
	graph *graph_d 
		= new graph	(json_file); 
	
	cout<<"GPU  NUMBER = "<<GPU_NUM<<endl;
	cout<<"PART NUMBER = "<<PART_NUM<<endl;

	mygraph=graph_d;
	cout<<"rank by degree\n";
double tt0=wtime();
//	graph_d->rank_by_degree();
//	graph_d->partition();
	graph_d->preproc();
double tt1=wtime();
cout<<"pre-processing time = "<<tt1-tt0<<endl;

	double total_t=0;
int r=5;
for(int n=0; n<r; n++){
	double t0=wtime();
	index_t total = 0;
	for(int i=0; i<PART_NUM; i++){
		graph_d->chunk_proc[i]=0;
	}

//int tid;
omp_set_nested(1);
{
#pragma omp parallel for num_threads(GPU_NUM+1)
//#pragma omp parallel for num_threads(GPU_NUM)
//	for(int i=0; i<GPU_NUM; i++){
	for(int i=0; i<GPU_NUM+1; i++){
//		tid = omp_get_thread_num();
		if(i<GPU_NUM){
			graph_d->gpuProc(i);
		}
		else if(i == GPU_NUM){
      			graph_d->cpuProc();
		}
		else cout<<"tid = "<<i<<endl;
	}
}

	
	for(int i=0; i<GPU_NUM+1; i++){
		total+= graph_d->count[i];
	}
	double t1=wtime();
	cout<<"total count "<<total<<"\n";
	cout<<"total time  "<<t1-t0<<" seconds\n";
	total_t += t1-t0;
}
/*
int r=5;
for(int n=0; n<r; n++){
	double t0=wtime();
	index_t total = 0;
#pragma omp parallel for num_threads(GPU_NUM)
	for(int i=0; i<GPU_NUM;i++){
		part_scan(i);
	}
	
	for(int i=0;i<GPU_NUM;i++){
		total+= graph_d->count[i];
	}
	double t1=wtime();
	cout<<"total count "<<total<<"\n";
	cout<<"total time  "<<t1-t0<<" seconds\n";
	total_t += t1-t0;
}
*/
cout<<"merge average time of 5 round = "<<total_t/r<<endl;
	return 0;
}
