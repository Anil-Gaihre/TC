#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 
#define thread_limit 256 
#define block_limit 1024 

#define GPU_COWORKER 1 
graph * mygraph;

__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];


		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/max_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + max_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/max_thd;
			top = top*m/max_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += max_thd;
		
		}
		tid += GPU_COWORKER * gridDim.x*blockDim.x/ max_thd;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += GPU_COWORKER* blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();

}

//----------------------------------------------------------------------------------------

__global__ void classify_kernel	//step 1: classify the edge list into different arrays
(	vertex_t* adj_list,
	vertex_t* head_list,
	index_t* begin,
	index_t  N,		//inputs
	index_t* small_num,
	index_t* mid_num,
	index_t* large_num
	//outputs: small/large head, adjacent, and number by thread
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space
	index_t small_offset=0;
	index_t mid_offset=0;
	index_t large_offset=0;
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
//			else{
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				large_offset++;
			}
		}
	}
	small_num[tid] = small_offset;
	mid_num[tid]   = mid_offset;
	large_num[tid] = large_offset;

}

__global__ void prefix_kernel_1	//this prefix scan function could be easier for data size is always 256*256
(	
 	index_t*	data,
	index_t*	block_offset
)
{
		
	//step 1: each block do prefix sum inside
	int tid = threadIdx.x +blockIdx.x*blockDim.x;

	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = data[tid];
	__syncthreads();

	index_t val=0;
	for(int i=0; i<=threadIdx.x; i++){
		val += temp_in[i];
	}


	__syncthreads();
	
	if(threadIdx.x==255){
		block_offset[blockIdx.x] = val;
		
	}
	data[tid] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_2	
(	
	index_t*	block_offset
)
{
	//step 2: collect each block's offset and do prefix for this set
	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = block_offset[threadIdx.x];
	__syncthreads();
	index_t val=0;
	for(int i=0; i<threadIdx.x; i++){
		val += temp_in[i];
	}
//		val = temp_in[threadIdx.x];
	block_offset[threadIdx.x] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_3	
(	
	index_t*	data,
	index_t*	block_offset
)
{
	//step 3: update by adding block offset
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	index_t val = data[tid];
	index_t offset = block_offset[blockIdx.x];
	val += offset;

	data[tid] = val;
	__syncthreads();
}

__global__ void collect_kernel
(	vertex_t* 	adj_list,
	vertex_t* 	head_list,
	index_t* 	begin,
	index_t	N,
	index_t* 	small_num,
	index_t* 	mid_num,
	index_t* 	large_num,
	index_t 	N1,
	index_t	N2,
	vertex_t*	dest_head,
	vertex_t*	dest_adj
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space


	index_t thd_base_small = 0;
	index_t thd_base_mid   = N1;
	index_t thd_base_large = N1+N2;
	if(tid!=0){
		thd_base_small = small_num[tid-1];
		thd_base_mid   = N1 + mid_num[tid-1];
		thd_base_large = N1 + N2 + large_num[tid-1];
	}
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	index_t small_offset = thd_base_small;
	index_t mid_offset   = thd_base_mid;
	index_t large_offset = thd_base_large;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				dest_head[small_offset] = head;
				dest_adj [small_offset] = adj;
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
//			else{
				dest_head[mid_offset] = head;
				dest_adj [mid_offset] = adj;
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				dest_head[large_offset] = head;
				dest_adj [large_offset] = adj;
				large_offset++;
			}
		}
	}
}


__global__ void reduce_kernel2(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void graph::scan(){

	hipSetDevice(4);
	vertex_t*	dev_adj;
	vertex_t*	dev_head;
	index_t*	dev_begin;
	index_t*	dev_count;

	H_ERR(hipMalloc(&dev_adj, upperEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_head, upperEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );
	
	index_t* block_offset;
	H_ERR(hipMalloc(&block_offset, max_block*sizeof(index_t)) );
	
	vertex_t*	classified_head;
	vertex_t*	classified_adj;
	
	index_t*	small_num;
	index_t*	mid_num;
	index_t*	large_num;

	vertex_t*	src_head;
	vertex_t*	src_adj;
	
	H_ERR(hipMalloc(&small_num, max_thd*max_block*sizeof(index_t)) );
	H_ERR(hipMalloc(&mid_num,   max_thd*max_block*sizeof(index_t)) );
	H_ERR(hipMalloc(&large_num, max_thd*max_block*sizeof(index_t)) );
	H_ERR(hipMalloc(&src_head, upperEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&src_adj,  upperEdgeCount*sizeof(vertex_t)) );
	
	H_ERR(hipMemcpy(src_adj,    upperAdj, upperEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(src_head,   upperHead, upperEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,   upperBegin, (vert_count+1)*sizeof(index_t), hipMemcpyHostToDevice) );

	dev_adj = src_adj;
	dev_head= src_head;
//		H_ERR(hipMemcpy(src_degree, degree, vert_count*sizeof(index_t), hipMemcpyHostToDevice) );
	
	H_ERR(hipMalloc(&classified_head, upperEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&classified_adj,  upperEdgeCount*sizeof(vertex_t)) );
	//

double time1=wtime();
	H_ERR(hipDeviceSynchronize() );

	
	classify_kernel <<<max_block,max_thd>>>(
				src_adj,
				src_head,
				dev_begin,
				upperEdgeCount,
				small_num,
				mid_num,
				large_num
				);
	H_ERR(hipDeviceSynchronize() );

	//test for prefix sum

	prefix_kernel_1 <<<max_block,max_thd>>>(small_num, block_offset);
	H_ERR(hipDeviceSynchronize() );
	prefix_kernel_2 <<<1,max_thd>>>(block_offset);
	H_ERR(hipDeviceSynchronize() );
	prefix_kernel_3 <<<max_block,max_thd>>>(small_num, block_offset);
	H_ERR(hipDeviceSynchronize() );

	prefix_kernel_1 <<<max_block,max_thd>>>(mid_num, block_offset);
	H_ERR(hipDeviceSynchronize() );
	prefix_kernel_2 <<<1,max_thd>>>(block_offset);
	H_ERR(hipDeviceSynchronize() );
	prefix_kernel_3 <<<max_block,max_thd>>>(mid_num, block_offset);
	H_ERR(hipDeviceSynchronize() );
	
	prefix_kernel_1 <<<max_block,max_thd>>>(large_num, block_offset);
	H_ERR(hipDeviceSynchronize() );
	prefix_kernel_2 <<<1,max_thd>>>(block_offset);
	H_ERR(hipDeviceSynchronize() );
	prefix_kernel_3 <<<max_block,max_thd>>>(large_num, block_offset);
	H_ERR(hipDeviceSynchronize() );

index_t N1,N2,N3;	
	H_ERR(hipMemcpy(&N1 ,  &small_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
	H_ERR(hipMemcpy(&N2 , &mid_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
	H_ERR(hipMemcpy(&N3 ,  &large_num[65535]   , sizeof(index_t), hipMemcpyDeviceToHost) );

	H_ERR(hipDeviceSynchronize() );
//	cout<<"N1 = "<<N1<<"\n";
//	cout<<"N2 = "<<N2<<"\n";
//	cout<<"N3 = "<<N3<<"\n";
	
	collect_kernel <<<max_block,max_thd>>>(
				src_adj,
				src_head,
				dev_begin,
				upperEdgeCount,
				small_num,
				mid_num,
				large_num,
				N1,
				N2,
				classified_head,
				classified_adj
				);
	H_ERR(hipDeviceSynchronize() );


double time2=wtime();


	warp_binary_kernel<<<max_block,max_thd>>>
	(	classified_head,
		classified_adj,
		dev_adj,
//			dev_degree,
		dev_begin,
		0,
		N1,
		dev_count
	);
	H_ERR(hipDeviceSynchronize() );
	block_binary_kernel<<<max_block,max_thd>>>
	(	classified_head,
		classified_adj,
		//dev_head,
		//dev_adj,

		dev_adj,
//			dev_degree,
		dev_begin,
		N1,
		N1+N2,
//		upperEdgeCount,
		dev_count
	);
	H_ERR(hipDeviceSynchronize() );
	
	reduce_kernel2 <<<1,1>>>(dev_count);
	H_ERR(hipDeviceSynchronize() );
	
	H_ERR(hipMemcpy(&count[0], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));

double time4 = wtime();
	cout<<"total count = "<<count[0]<<endl;
	cout<<"GPU time = "<<time4-time2<<" seconds"<<endl;
	
	H_ERR(hipFree(small_num) );
	H_ERR(hipFree(large_num) );
	H_ERR(hipFree(classified_head) );
	H_ERR(hipFree(classified_adj) );
	H_ERR(hipFree(src_head) );
	H_ERR(hipFree(src_adj) );

	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(block_offset) );
	H_ERR(hipFree(dev_count) );
}


