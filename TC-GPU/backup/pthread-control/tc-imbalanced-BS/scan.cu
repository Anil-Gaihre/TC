#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 
#define thread_limit 256 
#define block_limit 1024 

#define GPU_COWORKER 1 
graph * mygraph;

__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];


		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/max_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + max_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/max_thd;
			top = top*m/max_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += max_thd;
		
		}
		tid += GPU_COWORKER * gridDim.x*blockDim.x/ max_thd;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += GPU_COWORKER* blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();

}

//----------------------------------------------------------------------------------------

__global__ void classify_kernel	//step 1: classify the edge list into different arrays
(	vertex_t* adj_list,
	vertex_t* head_list,
	index_t* begin,
	index_t  N,		//inputs
	index_t* small_num,
	index_t* mid_num,
	index_t* large_num
	//outputs: small/large head, adjacent, and number by thread
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space
	index_t small_offset=0;
	index_t mid_offset=0;
	index_t large_offset=0;
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				large_offset++;
			}
		}
	}
	small_num[tid] = small_offset;
	mid_num[tid]   = mid_offset;
	large_num[tid] = large_offset;

}

__global__ void prefix_kernel_1	//this prefix scan function could be easier for data size is always 256*256
(	
 	index_t*	data,
	index_t*	block_offset
)
{
		
	//step 1: each block do prefix sum inside
	int tid = threadIdx.x +blockIdx.x*blockDim.x;

	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = data[tid];
	__syncthreads();

	index_t val=0;
	for(int i=0; i<=threadIdx.x; i++){
		val += temp_in[i];
	}


	__syncthreads();
	
	if(threadIdx.x==255){
		block_offset[blockIdx.x] = val;
		
	}
	data[tid] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_2	
(	
	index_t*	block_offset
)
{
	//step 2: collect each block's offset and do prefix for this set
	__shared__ index_t temp_in[256];
	temp_in[threadIdx.x] = block_offset[threadIdx.x];
	__syncthreads();
	index_t val=0;
	for(int i=0; i<threadIdx.x; i++){
		val += temp_in[i];
	}
//		val = temp_in[threadIdx.x];
	block_offset[threadIdx.x] = val;
	__syncthreads();
	
}

__global__ void prefix_kernel_3	
(	
	index_t*	data,
	index_t*	block_offset
)
{
	//step 3: update by adding block offset
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	index_t val = data[tid];
	index_t offset = block_offset[blockIdx.x];
	val += offset;

	data[tid] = val;
	__syncthreads();
}

__global__ void collect_kernel
(	vertex_t* 	adj_list,
	vertex_t* 	head_list,
	index_t* 	begin,
	index_t	N,
	index_t* 	small_num,
	index_t* 	mid_num,
	index_t* 	large_num,
	index_t 	N1,
	index_t	N2,
	vertex_t*	dest_head,
	vertex_t*	dest_adj
)
{
	int tid = threadIdx.x +blockIdx.x*blockDim.x;
	index_t bin_size = (N-1)/(blockDim.x*gridDim.x)+1;
	index_t thd_base = tid*bin_size;		//start point of threads space


	index_t thd_base_small = 0;
	index_t thd_base_mid   = N1;
	index_t thd_base_large = N1+N2;
	if(tid!=0){
		thd_base_small = small_num[tid-1];
		thd_base_mid   = N1 + mid_num[tid-1];
		thd_base_large = N1 + N2 + large_num[tid-1];
	}
	
	//temp variables
	vertex_t head;
	vertex_t adj;
	index_t m;
	index_t n;
	index_t small_offset = thd_base_small;
	index_t mid_offset   = thd_base_mid;
	index_t large_offset = thd_base_large;
	for(index_t i=0; i<bin_size; i++){
		index_t id = thd_base + i;
		if(id<N){
			head = head_list[id];
			adj  = adj_list[id];
			m = begin[head+1]-begin[head];//degree[head];
			n = begin[adj+1]-begin[adj];//degree[adj];
			if(m<n){
				n=m;
			}
			if(n<thread_limit && n>0){
				dest_head[small_offset] = head;
				dest_adj [small_offset] = adj;
				small_offset++;
			}
			else if(n>0){	//could be more then 2 catigories
				dest_head[mid_offset] = head;
				dest_adj [mid_offset] = adj;
				mid_offset++;
			}
			else {	//could be more then 2 catigories
				dest_head[large_offset] = head;
				dest_adj [large_offset] = adj;
				large_offset++;
			}
		}
	}
}


__global__ void reduce_kernel2(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void* part_scan(void * data){

	index_t thd_count=0;	
	int GPU_id = *(int*)data;
	int i = GPU_id;
//	cout<<"GPU id = "<<GPU_id<<"\n";
	hipSetDevice(GPU_id);
	H_ERR(hipDeviceSynchronize() );

	vertex_t*	dev_adj;
	vertex_t*	dev_head;
	index_t*	dev_begin;
	index_t*	dev_count;

	index_t partEdgeCount = mygraph->partEdgeCount[i];
	vertex_t vert_count = mygraph->vert_count;
	vertex_t* partAdj = mygraph->partAdj[i];
	vertex_t* partHead= mygraph->partHead[i];
	index_t* partBegin  = mygraph->partBegin[i];
	index_t* count    = mygraph->count;

	H_ERR(hipMalloc(&dev_adj, partEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_head, partEdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

		
	
	index_t* block_offset;
	H_ERR(hipMalloc(&block_offset, max_block*sizeof(index_t)) );
	
	H_ERR(hipMemcpy(dev_adj,    partAdj, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_head,   partHead, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  partBegin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );


//---------------- go into loop-------------------
	double time2=wtime();
	for(int j=0; j<PART_NUM; j++){	

//---------------- check -------------------------
		if(mygraph->ds_status[i * GPU_NUM + j]>0){
			break;
		}
		mygraph->ds_status[i * GPU_NUM + j]=1;
		mygraph->ds_complete[i]++;
//---------------- run ---------------------------
		index_t totalEdgeCount = mygraph->partEdgeCount[j];
		vertex_t* 	head = mygraph->partHead[j];
		vertex_t* 	adj  = mygraph->partAdj[j];
		vertex_t*	classified_head;
		vertex_t*	classified_adj;
		
		index_t*	small_num;
		index_t*	mid_num;
		index_t*	large_num;

		vertex_t*	src_head;
		vertex_t*	src_adj;
		
		H_ERR(hipMalloc(&small_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&mid_num,   max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&large_num, max_thd*max_block*sizeof(index_t)) );
		H_ERR(hipMalloc(&src_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&src_adj,  totalEdgeCount*sizeof(vertex_t)) );
		
		H_ERR(hipMemcpy(src_adj,    adj, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(src_head,   head, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		
		H_ERR(hipMalloc(&classified_head, totalEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&classified_adj,  totalEdgeCount*sizeof(vertex_t)) );
		//

	double time1=wtime();
		H_ERR(hipDeviceSynchronize() );

		
		classify_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					dev_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num
					);
		H_ERR(hipDeviceSynchronize() );

		//test for prefix sum

		prefix_kernel_1 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(small_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

		prefix_kernel_1 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(mid_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		
		prefix_kernel_1 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_2 <<<1,max_thd>>>(block_offset);
		H_ERR(hipDeviceSynchronize() );
		prefix_kernel_3 <<<max_block,max_thd>>>(large_num, block_offset);
		H_ERR(hipDeviceSynchronize() );

	index_t N1,N2,N3;	
		H_ERR(hipMemcpy(&N1 ,  &small_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N2 , &mid_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
		H_ERR(hipMemcpy(&N3 ,  &large_num[65535]   , sizeof(index_t), hipMemcpyDeviceToHost) );

		H_ERR(hipDeviceSynchronize() );
	//	cout<<"N1 = "<<N1<<"\n";
	//	cout<<"N2 = "<<N2<<"\n";
	//	cout<<"N3 = "<<N3<<"\n";
		
		collect_kernel <<<max_block,max_thd>>>(
					src_adj,
					src_head,
					dev_begin,
					totalEdgeCount,
					small_num,
					mid_num,
					large_num,
					N1,
					N2,
					classified_head,
					classified_adj
					);
		H_ERR(hipDeviceSynchronize() );




		warp_binary_kernel<<<max_block,max_thd>>>
		(	classified_head,
			classified_adj,
			dev_adj,
			dev_begin,
			0,
			N1,
			dev_count
		);
		H_ERR(hipDeviceSynchronize() );

		block_binary_kernel<<<max_block,max_thd>>>
		(	classified_head,
			classified_adj,
			dev_adj,
			dev_begin,
			N1,
			N1+N2,//totalEdgeCount,
			dev_count
		);
		H_ERR(hipDeviceSynchronize() );
		
		reduce_kernel2 <<<1,1>>>(dev_count);
		H_ERR(hipDeviceSynchronize() );
		
		H_ERR(hipMemcpy(&count[i], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
		thd_count += count[i];
		
		H_ERR(hipFree(small_num) );
		H_ERR(hipFree(large_num) );
		H_ERR(hipFree(classified_head) );
		H_ERR(hipFree(classified_adj) );
		H_ERR(hipFree(src_head) );
		H_ERR(hipFree(src_adj) );
		cout<<"GPU "<<i<<" part "<<j<<"\n";

//---------------- write result ---------------------------
		mygraph->ds_count[i * GPU_NUM + j] = count[i];
	}
	
//---------------- work ending ---------------------------
//---------------- work stealing ---------------------------
	int check = 0;
	for(int k=0; k<GPU_NUM; k++){
		check += mygraph->ds_complete[k];
	}
	while(check<GPU_NUM*GPU_NUM){
//step 1: looking for the GPU with most remaining work
		int min=0;
		for(int k=GPU_NUM-1; k>=0; k--){
			if(mygraph->ds_complete[k]<mygraph->ds_complete[min]){
				min = k;
			}
			if(mygraph->ds_complete[k] == mygraph->ds_complete[min]){
				if(mygraph->ds_help[k] < mygraph->ds_help[min]){
					min = k;
				}
			}
		}
		mygraph->ds_help[min]++;

		if(mygraph->ds_complete[min]==GPU_NUM){
			for(int k=0; k<GPU_NUM; k++){
				check += mygraph->ds_complete[k];
			}
			continue;
		}

//step 2: check ds_complete array and start moving data
		H_ERR(hipFree(dev_adj) );
		H_ERR(hipFree(dev_head) );

		index_t partEdgeCount = mygraph->partEdgeCount[min];
		partAdj = mygraph->partAdj[min];
		partHead= mygraph->partHead[min];
		partBegin  = mygraph->partBegin[min];

		H_ERR(hipMalloc(&dev_adj, partEdgeCount*sizeof(vertex_t)) );
		H_ERR(hipMalloc(&dev_head, partEdgeCount*sizeof(vertex_t)) );

		
		H_ERR(hipMemcpy(dev_adj,    partAdj, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(dev_head,   partHead, partEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
		H_ERR(hipMemcpy(dev_begin,  partBegin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );
		

//step 3: set flags and work
		int j = GPU_NUM - 1;
		while(j>=0){
			if(mygraph->ds_status[min * GPU_NUM + j]>0){
				j--;
				continue;
			}
			mygraph->ds_status[min * GPU_NUM + j] = 1;
			mygraph->ds_complete[min]++;
	//work		
			index_t totalEdgeCount = mygraph->partEdgeCount[j];
			vertex_t* 	head = mygraph->partHead[j];
			vertex_t* 	adj  = mygraph->partAdj[j];
			vertex_t*	classified_head;
			vertex_t*	classified_adj;
		
			index_t*	small_num;
			index_t*	mid_num;
			index_t*	large_num;

			vertex_t*	src_head;
			vertex_t*	src_adj;
		
			H_ERR(hipMalloc(&small_num, max_thd*max_block*sizeof(index_t)) );
			H_ERR(hipMalloc(&mid_num,   max_thd*max_block*sizeof(index_t)) );
			H_ERR(hipMalloc(&large_num, max_thd*max_block*sizeof(index_t)) );
			H_ERR(hipMalloc(&src_head, totalEdgeCount*sizeof(vertex_t)) );
			H_ERR(hipMalloc(&src_adj,  totalEdgeCount*sizeof(vertex_t)) );
		
			H_ERR(hipMemcpy(src_adj,    adj, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
			H_ERR(hipMemcpy(src_head,   head, totalEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
			
			H_ERR(hipMalloc(&classified_head, totalEdgeCount*sizeof(vertex_t)) );
			H_ERR(hipMalloc(&classified_adj,  totalEdgeCount*sizeof(vertex_t)) );
		//

	double time1=wtime();
			H_ERR(hipDeviceSynchronize() );

			
			classify_kernel <<<max_block,max_thd>>>(
						src_adj,
						src_head,
						dev_begin,
						totalEdgeCount,
						small_num,
						mid_num,
						large_num
						);
			H_ERR(hipDeviceSynchronize() );

			//test for prefix sum

			prefix_kernel_1 <<<max_block,max_thd>>>(small_num, block_offset);
			H_ERR(hipDeviceSynchronize() );
			prefix_kernel_2 <<<1,max_thd>>>(block_offset);
			H_ERR(hipDeviceSynchronize() );
			prefix_kernel_3 <<<max_block,max_thd>>>(small_num, block_offset);
			H_ERR(hipDeviceSynchronize() );

			prefix_kernel_1 <<<max_block,max_thd>>>(mid_num, block_offset);
			H_ERR(hipDeviceSynchronize() );
			prefix_kernel_2 <<<1,max_thd>>>(block_offset);
			H_ERR(hipDeviceSynchronize() );
			prefix_kernel_3 <<<max_block,max_thd>>>(mid_num, block_offset);
			H_ERR(hipDeviceSynchronize() );
			
			prefix_kernel_1 <<<max_block,max_thd>>>(large_num, block_offset);
			H_ERR(hipDeviceSynchronize() );
			prefix_kernel_2 <<<1,max_thd>>>(block_offset);
			H_ERR(hipDeviceSynchronize() );
			prefix_kernel_3 <<<max_block,max_thd>>>(large_num, block_offset);
			H_ERR(hipDeviceSynchronize() );

			index_t N1,N2,N3;	
			H_ERR(hipMemcpy(&N1 ,  &small_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
			H_ERR(hipMemcpy(&N2 , &mid_num[65535] , sizeof(index_t), hipMemcpyDeviceToHost) );
			H_ERR(hipMemcpy(&N3 ,  &large_num[65535]   , sizeof(index_t), hipMemcpyDeviceToHost) );

			H_ERR(hipDeviceSynchronize() );
		
			collect_kernel <<<max_block,max_thd>>>(
						src_adj,
						src_head,
						dev_begin,
						totalEdgeCount,
						small_num,
						mid_num,
						large_num,
						N1,
						N2,
						classified_head,
						classified_adj
						);
			H_ERR(hipDeviceSynchronize() );




			warp_binary_kernel<<<max_block,max_thd>>>
			(	classified_head,
				classified_adj,
				dev_adj,
				dev_begin,
				0,
				N1,
				dev_count
			);
			H_ERR(hipDeviceSynchronize() );

			block_binary_kernel<<<max_block,max_thd>>>
			(	classified_head,
				classified_adj,
				dev_adj,
				dev_begin,
				N1,
				N1+N2,
				dev_count
			);
			H_ERR(hipDeviceSynchronize() );
			
			reduce_kernel2 <<<1,1>>>(dev_count);
			H_ERR(hipDeviceSynchronize() );
			
			H_ERR(hipMemcpy(&count[min], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
//---------------- write result ---------------------------
			mygraph->ds_count[min * GPU_NUM + j] = count[min];
			thd_count += count[min];
			
			H_ERR(hipFree(small_num) );
			H_ERR(hipFree(large_num) );
			H_ERR(hipFree(classified_head) );
			H_ERR(hipFree(classified_adj) );
			H_ERR(hipFree(src_head) );
			H_ERR(hipFree(src_adj) );
			cout<<"steal GPU "<<i<<" GPU "<<min<<" part "<<j<<"\n";


		}
//step 4: write result
		for(int k=0; k<GPU_NUM; k++){
			check += mygraph->ds_complete[k];
		}
	}


//---------------- work stealing end -----------------------

	double time4 = wtime();
	count[i] = thd_count;
	cout<<"gpu "<<i<<" binary count="<<count[i]<<"\n";
	cout<<"time = "<<time4-time2<<" seconds"<<endl;
	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_head) );
	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(block_offset) );
	H_ERR(hipFree(dev_count) );
	return NULL;	
}


