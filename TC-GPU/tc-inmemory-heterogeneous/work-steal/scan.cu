#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include <stdio.h>
#include "iostream"
#define max_thd 256 
#define max_block 256 

graph * mygraph;
__global__ void block_binary_kernel
(	//vertex_t*	head,
	//vertex_t*	adj,
	Edge*		workload,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
//		vertex_t A = head[tid];
//		vertex_t B = adj[tid];
		vertex_t A = workload[tid].A;
		vertex_t B = workload[tid].B;
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];


		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/max_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
//printf("find A %d B %d C %d\n",A,B,X);
					mycount++;
					bot = top + max_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/max_thd;
			top = top*m/max_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += max_thd;
		
		}
		tid += GPU_PER_PART * gridDim.x*blockDim.x/256;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
//		count[blockIdx.x]+=val;
		count[blockIdx.x]=val;
//		if(val!=0)
//			printf("+ %d\n",count[blockIdx.x]);
	}
}

__global__ void warp_binary_kernel
(	//vertex_t*	head,
	//vertex_t*	adj,
	Edge*		workload,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = workload[tid].A;
		vertex_t B = workload[tid].B;
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];
//if(i==0) printf("A %d B %d\n");
		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
//		tid += GPU_NUM* blockDim.x*gridDim.x/32;
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
//		count[blockIdx.x]=val;
		count[blockIdx.x]+=val;
	}
	__syncthreads();

}


__global__ void init_count(index_t* count)
{
	int tid = threadIdx.x;
	count[tid] = 0;
}

__global__ void reduce_kernel(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}


//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------



void graph::initDevice(int GPU_id,int Part_id){
//cuda memory copy of partAdj and partBegin
	hipSetDevice(GPU_id);

	int P=Part_id;
	H_ERR(hipDeviceSynchronize() );


	vertex_t*	dev_adj;		
	index_t*	dev_begin;	
	index_t*	dev_count;	
	Edge*		buffer0;	
	Edge*		buffer1;	

	index_t EdgeCount = partEdgeCount[P];
	vertex_t* Adj = partAdj[P];
	index_t* Begin  = partBegin[P];
	
	H_ERR(hipMalloc(&dev_adj, EdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

	H_ERR(hipMemcpy(dev_adj,    Adj, EdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  Begin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );
	
	H_ERR(hipMalloc(&buffer0,    BufferSize*sizeof(Edge)) );
	H_ERR(hipMalloc(&buffer1,    BufferSize*sizeof(Edge)) );
	
	gdata[GPU_id].adj	=	dev_adj;
	gdata[GPU_id].begin	=	dev_begin;
	gdata[GPU_id].count	=	dev_count;
	gdata[GPU_id].EdgeBuffer[0]=	buffer0;
	gdata[GPU_id].EdgeBuffer[1]=	buffer1;
	gdata[GPU_id].partition_id =	P;
	gdata[GPU_id].currentBuffer=	0;
	init_count <<<1,max_thd>>>(dev_count);

}

void graph::DeviceCompute(int GPU_id, index_t Chunk_id){
	
	int P = gdata[GPU_id].partition_id;
	
	vertex_t*	dev_adj		=gdata[GPU_id].adj;
	index_t*	dev_begin	=gdata[GPU_id].begin;
	index_t*	dev_count	=gdata[GPU_id].count;
	Edge*		buffer		=gdata[GPU_id].EdgeBuffer[gdata[GPU_id].currentBuffer%2];
	gdata[GPU_id].currentBuffer	=1-gdata[GPU_id].currentBuffer;
	index_t currentBufferSize = BufferSize;
	if(Chunk_id==upperEdgeCount/BufferSize){
		currentBufferSize = upperEdgeCount % BufferSize;
	}
	init_count <<<1,max_thd>>>(dev_count);
	H_ERR(hipMemcpy(buffer, &OrientedEdge[Chunk_id*BufferSize], currentBufferSize*sizeof(Edge), hipMemcpyHostToDevice) );
	H_ERR(hipDeviceSynchronize() );
	warp_binary_kernel<<<max_block,max_thd>>>
	(	buffer,
		dev_adj,
		dev_begin,
		0,
		currentBufferSize,
		dev_count
	);
	//write the result of this chunk back
	H_ERR(hipDeviceSynchronize() );
	index_t tempcount[max_block];
	index_t mycount=0;
	H_ERR(hipMemcpy(tempcount, dev_count, max_block*sizeof(index_t), hipMemcpyDeviceToHost));
	for(int i=0; i<max_block; i++) mycount += tempcount[i];
	ds_count[P * ChunkNum + Chunk_id] = mycount;
}

void graph::gpuReduce(int GPU_id){
	vertex_t*	dev_adj		=gdata[GPU_id].adj;
	index_t*	dev_begin	=gdata[GPU_id].begin;
	index_t*	dev_count	=gdata[GPU_id].count;
	Edge**		buffer		=gdata[GPU_id].EdgeBuffer;
	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_begin) );
	H_ERR(hipFree(dev_count) );
	H_ERR(hipFree(buffer[0]) );
	H_ERR(hipFree(buffer[1]) );
}

void graph::gpuProc(int GPU_id){
double t0 = wtime();
	

//	step 1: computing ------------------------------------------------------------------------

	bool STOP = 0;
	for(int P=0; P<PART_NUM; P++){
		if(STOP){break;}
		initDevice(GPU_id,P);
		for(index_t i=GPU_id; i<ChunkNum; i+=DEV_NUM ){
			if(ds_status[P*ChunkNum + i]!=0){
			//finish with someone's help
				STOP =1;

//				ds_complete[GPU_id] = ds_last[GPU_id];
				break;
			}
			//else
			ds_status[P*ChunkNum + i] = 1;
			ds_complete[GPU_id]++;
			DeviceCompute(GPU_id,i);
//cout<<"GPU "<<GPU_id<<" chunk "<<i<<endl;

		}
		gpuReduce(GPU_id);
	}

//step 2: work stealing-----------------------------------------------------------------------------
	
	index_t check = 0;
	for(int k=0; k<DEV_NUM; k++){
		check += ds_complete[k];
//cout<<"device "<<k<<" complete "<<ds_complete[k]<<endl;
	}


	while (check< PART_NUM*ChunkNum){//while()
	//step 2-1: looking for the GPU with most remaining work
		if(STOP){
			break;
		}
		int MIN=GPU_id;
		for(int k=DEV_NUM-1; k>=0; k--){
			if(ds_complete[k] < ds_complete[MIN]){
				if(ds_help[k]==0){
					MIN = k;
					ds_help[MIN] = 1;
				}
			}
		}
	//step 2-2: help this device from i
		
		index_t i = MIN + (ds_last[MIN]-1)*DEV_NUM + (PART_NUM-1)*ChunkNum;
		//find the chunk id=i that start from the end to help
		if(MIN == GPU_id){
			STOP = 1;
			break;
		}
//cout<<"GPU "<<GPU_id<<" steal work from device "<<MIN<<endl;
//cout<<"ds last "<<ds_last[MIN]<<endl;
//cout<<"GPU "<<GPU_id<<" steal work from chunk id "<<i<<endl;
		
//		initDevice(GPU_id,PART_NUM-1);
		//for in memory version, we don't need to initiate for the only part again
	
		while(i>0){
			
			int P = i / ChunkNum;
			int j = i % ChunkNum;
			
			if(ds_status[i]!=0) {
//				help = 1;//set for outer-loop
//cout<<"Whoops finised help"<<endl;			
//				ds_complete[MIN] = ds_last[MIN];
				STOP = 1;
				break;
//				return;
			}
			//finish with someone's help
			//
			ds_status[i] = 1; // i = P*ChunkNum + j;
			ds_complete[MIN]++;
//cout<<"help ++ Part: "<<P<<"; chunk: "<<j<<endl;				
			DeviceCompute(GPU_id,j);
//cout<<"done"<<endl;
			//set next i
			if(j>=DEV_NUM){
				i -= DEV_NUM;
			}
			else{
//cout<<"jump partition"<<endl;
				i = MIN + (ds_last[MIN]-1)*DEV_NUM + (P-1)*ChunkNum;
				if(i<0) break;

				if(ds_status[i]==0) {
					gpuReduce(GPU_id);
					initDevice(GPU_id,P-1);
				}
			}	
			
		}
//helped one device, continue

		check = 0;
		for(int k=0; k<DEV_NUM; k++){
			check += ds_complete[k];
//cout<<"device "<<k<<" complete "<<ds_complete[k]<<endl;
		}
	}


	
// work stealing done-------------------------------------------------------------------------

//cout<<"GPU "<<GPU_id<<" finished all  jobs"<<endl;
//	count[GPU_id] = total_count;
double t1 = wtime();
	gpuReduce(GPU_id);//for one partition only
cout<<"GPU "<<GPU_id<<" time = "<<t1-t0<<endl;
//cout<<"GPU complete = "<<ds_complete[GPU_id];
}
