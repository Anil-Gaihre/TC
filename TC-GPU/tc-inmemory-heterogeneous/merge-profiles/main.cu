#include "hip/hip_runtime.h"
//Graph format: Json based format
//Storage format: 
//struct{
//		int: src_ver
//		Arr: [ver_0|ver_1|ver_2|...]
//		Int: num_conn_ver
//	}
/* main.cu */
#include "graph.h"
#include <sstream>
#include <iostream>
#include <fstream>
#include <pthread.h>
#define N 256*256
using namespace std;



int main(int args, char *argv[]) {
//	pthread_t thd1;
	std::cout<<"Input format: ./exe graph-file-name"
						<<" (json formated file)\n";

	if(args != 2) return -1;
	string json_file 	= argv[1];
	graph *graph_d 
		= new graph	(json_file); 
	
	cout<<"GPU  NUMBER = "<<GPU_NUM<<endl;
	cout<<"PART NUMBER = "<<PART_NUM<<endl;

	cout<<"rank by degree\n";
double tt0=wtime();
	graph_d->preproc();
double tt1=wtime();
cout<<"pre-processing time = "<<tt1-tt0<<endl;

	double total_t=0;
int r=2;
for(int n=0; n<r; n++){
	double t0=wtime();
//	index_t total = 0;

//int tid;
	graph_d->gpuProc(0);

	
//	for(int i=0; i<GPU_NUM+1; i++){
//		total+= graph_d->count[i];
//	}
	graph_d->reduceResult();
	double t1=wtime();
	cout<<"total count "<<graph_d->count[0]<<"\n";
	cout<<"total time  "<<t1-t0<<" seconds\n";
	total_t += t1-t0;
}
cout<<"merge average time of 5 round = "<<total_t/r<<endl;
	return 0;
}
