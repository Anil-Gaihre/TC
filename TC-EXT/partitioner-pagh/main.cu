//Graph format: Json based format
//Storage format: 
//struct{
//		int: src_ver
//		Arr: [ver_0|ver_1|ver_2|...]
//		Int: num_conn_ver
//	}
/* main.cu */
#include "graph.h"
#include <sstream>
#include <iostream>
#include <fstream>
#include <pthread.h>
#define N 256*256
using namespace std;



int main(int args, char *argv[]) {
//	pthread_t thd1;
	std::cout<<"Input format: ./exe graph-file-name"
						<<" (json formated file)\n";

	if(args != 2) return -1;
	string json_file 	= argv[1];
	graph *graph_d 
		= new graph	(json_file); 
	
	cout<<"GPU  NUMBER = "<<GPU_NUM<<endl;
	cout<<"PART NUMBER = "<<PART_NUM<<endl;

	cout<<"rank by degree\n";
	
	graph_d->preproc();
	graph_d->edge_2d();
	graph_d->write_back();

	return 0;
}
