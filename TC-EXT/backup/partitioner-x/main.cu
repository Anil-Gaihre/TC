#include "hip/hip_runtime.h"
//Graph format: Json based format
//Storage format: 
//struct{
//		int: src_ver
//		Arr: [ver_0|ver_1|ver_2|...]
//		Int: num_conn_ver
//	}
/* main.cu */
#include "graph.h"
#include <sstream>
#include <iostream>
#include <fstream>
#include <pthread.h>
#define N 256*256
using namespace std;



int main(int args, char *argv[]) {
//	pthread_t thd1;
	std::cout<<"Input format: ./exe graph-file-name"
						<<" (json formated file)\n";

	if(args != 2) return -1;
	string json_file 	= argv[1];
	graph *graph_d 
		= new graph	(json_file); 

	PART_NUM = 1 + (int)sqrt((graph_d->edge_count-1)/PartitionSize);
//	PART_NUM = (int)sqrt((graph_d->edge_count-1)/PartitionSize);
	cout<<"PART NUMBER = "<<PART_NUM<<endl;

	cout<<"rank by degree\n";
	
	graph_d->vertical_partition();
	
//	graph_d->part_validation();

	graph_d->further_partition();
	graph_d->edge_2d();

//	graph_d->write_back();

	return 0;
}
