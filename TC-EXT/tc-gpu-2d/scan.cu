#include "hip/hip_runtime.h"
#include "cuUtil.cu"
#include "scan.h"
#include "comm.h"
#include "wtime.h"
#include "graph.h"
#include "worker.h"
#include <stdio.h>
#include "iostream"
#define max_thd 256 
#define max_block 256
using namespace std;

__global__ void warp_binary_kernel
(
	//input data
	vertex_t	*adj1,
	index_t		*beg1,
	vertex_t	*adj2,
	index_t		*beg2,
	Edge*		buffer,
	//parameters
	index_t		bufferSize,
	vertex_t	offsetA,
	vertex_t	offsetB,
	//output
	index_t*	count	
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<bufferSize){
		vertex_t A = buffer[tid].A - offsetA;
		vertex_t B = buffer[tid].B - offsetB;
		index_t m = beg1[A+1]-beg1[A];//degree[A];
		index_t n = beg2[B+1]-beg2[B];//degree[B];
//if(i==0) printf("A %d B %d\n");
		vertex_t* a = &(adj1[beg1[A]]);
		vertex_t* b = &(adj2[beg2[B]]);
		
		index_t tempd;
		vertex_t *tempa;	
		if(m<n){
			tempa = a;
			a = b;
			b = tempa;
			tempd = m;
			m = n;
			n = tempd;
		}


		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
//		tid += GPU_NUM* blockDim.x*gridDim.x/32;
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
//		count[blockIdx.x]+=val;
	}
	__syncthreads();

}

/*
__global__ void warp_binary_kernel
(	//vertex_t*	head,
	//vertex_t*	adj,
	Edge*		workload,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = workload[tid].A;
		vertex_t B = workload[tid].B;
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];
//if(i==0) printf("A %d B %d\n");
		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
//		tid += GPU_NUM* blockDim.x*gridDim.x/32;
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
//		count[blockIdx.x]=val;
		count[blockIdx.x]+=val;
	}
	__syncthreads();

}
*/

__global__ void init_count(index_t* count)
{
	int tid = threadIdx.x;
	count[tid] = 0;
}

__global__ void reduce_kernel(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}


//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void worker::init_gpu(){

	H_ERR(hipSetDevice(1));

	H_ERR(hipMalloc(&gdata.adj1, csr1->adj_size*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&gdata.beg1, csr1->beg_size*sizeof(index_t)) );

	H_ERR(hipMalloc(&gdata.adj2, csr2->adj_size*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&gdata.beg2, csr2->beg_size*sizeof(index_t)) );
	
	H_ERR(hipMalloc(&gdata.buffer, BufferSize*sizeof(Edge)) );
	H_ERR(hipMalloc(&gdata.count,    max_block*sizeof(index_t)) );
	
	H_ERR(hipDeviceSynchronize() );
	init_count <<<1,max_thd>>>(gdata.count);

	H_ERR(hipMemcpy(gdata.adj1, csr1->adj, csr1->adj_size*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(gdata.beg1, csr1->begin, csr1->beg_size*sizeof(index_t), hipMemcpyHostToDevice) );

	H_ERR(hipMemcpy(gdata.adj2, csr2->adj, csr2->adj_size*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(gdata.beg2, csr2->begin, csr2->beg_size*sizeof(index_t), hipMemcpyHostToDevice) );


	

}



void worker::call_gpu(){
	H_ERR(hipMemcpy(gdata.buffer, buffer->edge, buffer->chunk_size*sizeof(Edge), hipMemcpyHostToDevice) );
	H_ERR(hipDeviceSynchronize() );

//	cout<<"csr 1 id = "<<csr1->row_id<<", offsets 1: "<<offset[csr1->row_id]<<endl;
//	cout<<"csr 2 id = "<<csr2->row_id<<", offsets 2: "<<offset[csr2->row_id]<<endl;
	//compute
	warp_binary_kernel<<<max_block,max_thd>>>
	(	
		gdata.adj1,	
		gdata.beg1,	
		gdata.adj2,	
		gdata.beg2,	
		gdata.buffer,
		buffer->chunk_size,
		offset[csr1->row_id],
		offset[csr2->row_id],
		gdata.count	
	);

	//write the result of this chunk back
	H_ERR(hipDeviceSynchronize() );
	index_t tempcount[max_block];
	index_t mycount=0;
	H_ERR(hipMemcpy(tempcount, gdata.count, max_block*sizeof(index_t), hipMemcpyDeviceToHost));
	for(int i=0; i<max_block; i++){ mycount += tempcount[i];}
	gdata.host_count = mycount;

}

void worker::free_gpu(){
	H_ERR(hipFree(gdata.adj1) );
	H_ERR(hipFree(gdata.beg1) );
	H_ERR(hipFree(gdata.adj2) );
	H_ERR(hipFree(gdata.beg2) );
	H_ERR(hipFree(gdata.buffer) );
	H_ERR(hipFree(gdata.count) );
}




