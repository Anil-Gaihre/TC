#include "hip/hip_runtime.h"
#include "cuUtil.cu"
#include "scan.h"
#include "comm.h"
#include "wtime.h"
#include "graph.h"
#include "worker.h"
#include <stdio.h>
#include "iostream"
#define max_thd 256 
#define max_block 256
using namespace std;

__global__ void warp_binary_kernel
(
	//input data
	vertex_t	*adj1,
	index_t		*beg1,
	vertex_t	*adj2,
	index_t		*beg2,
	Edge*		buffer,
	//parameters
	index_t		bufferSize,
	vertex_t	offsetA,
	vertex_t	offsetB,
	//output
	index_t*	count	
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<bufferSize){
		vertex_t A = buffer[tid].A - offsetA;
		vertex_t B = buffer[tid].B - offsetB;
		index_t m = beg1[A+1]-beg1[A];//degree[A];
		index_t n = beg2[B+1]-beg2[B];//degree[B];
//if(i==0) printf("A %d B %d\n");
		vertex_t* a = &(adj1[beg1[A]]);
		vertex_t* b = &(adj2[beg2[B]]);
		
		index_t tempd;
		vertex_t *tempa;	
		if(m<n){
			tempa = a;
			a = b;
			b = tempa;
			tempd = m;
			m = n;
			n = tempd;
		}

		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
//		tid += GPU_NUM* blockDim.x*gridDim.x/32;
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
//		count[blockIdx.x]+=val;
	}
	__syncthreads();

}

/*
__global__ void warp_binary_kernel
(	//vertex_t*	head,
	//vertex_t*	adj,
	Edge*		workload,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = workload[tid].A;
		vertex_t B = workload[tid].B;
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];
//if(i==0) printf("A %d B %d\n");
		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
//printf("find A %d B %d C %d\n",A,B,X);
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
//		tid += GPU_NUM* blockDim.x*gridDim.x/32;
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
//		count[blockIdx.x]=val;
		count[blockIdx.x]+=val;
	}
	__syncthreads();

}
*/

__global__ void init_count(index_t* count)
{
	int tid = threadIdx.x;
	count[tid] = 0;
}

__global__ void reduce_kernel(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<max_block; i++){
		val += count[i];
	}
	count[0] = val;
}


//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void worker::init_gpu(){

	H_ERR(hipMalloc(&gdata.adj1, csr1->adj_size*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&gdata.beg1, csr1->beg_size*sizeof(index_t)) );

	H_ERR(hipMalloc(&gdata.adj2, csr2->adj_size*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&gdata.beg2, csr2->beg_size*sizeof(index_t)) );
	
	H_ERR(hipMalloc(&gdata.buffer, BufferSize*sizeof(Edge)) );
	H_ERR(hipMalloc(&gdata.count,    max_block*sizeof(index_t)) );
	
	H_ERR(hipDeviceSynchronize() );
	init_count <<<1,max_thd>>>(gdata.count);

	H_ERR(hipMemcpy(gdata.adj1, csr1->adj, csr1->adj_size*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(gdata.beg1, csr1->begin, csr1->beg_size*sizeof(index_t), hipMemcpyHostToDevice) );

	H_ERR(hipMemcpy(gdata.adj2, csr2->adj, csr2->adj_size*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(gdata.beg2, csr2->begin, csr2->beg_size*sizeof(index_t), hipMemcpyHostToDevice) );


	

}



void worker::call_gpu(){
	H_ERR(hipMemcpy(gdata.buffer, buffer->edge, buffer->chunk_size*sizeof(Edge), hipMemcpyHostToDevice) );
	H_ERR(hipDeviceSynchronize() );

//	cout<<"csr 1 id = "<<csr1->row_id<<", offsets 1: "<<offset[csr1->row_id]<<endl;
//	cout<<"csr 2 id = "<<csr2->row_id<<", offsets 2: "<<offset[csr2->row_id]<<endl;
	//compute
	warp_binary_kernel<<<max_block,max_thd>>>
	(	
		gdata.adj1,	
		gdata.beg1,	
		gdata.adj2,	
		gdata.beg2,	
		gdata.buffer,
		buffer->chunk_size,
		offset[csr1->row_id],
		offset[csr2->row_id],
		gdata.count	
	);

	//write the result of this chunk back
	H_ERR(hipDeviceSynchronize() );
	index_t tempcount[max_block];
	index_t mycount=0;
	H_ERR(hipMemcpy(tempcount, gdata.count, max_block*sizeof(index_t), hipMemcpyDeviceToHost));
	for(int i=0; i<max_block; i++){ mycount += tempcount[i];}
	gdata.host_count = mycount;
	cout<<"gpu get result "<<mycount<<endl;

}

void worker::free_gpu(){
	H_ERR(hipFree(gdata.adj1) );
	H_ERR(hipFree(gdata.beg1) );
	H_ERR(hipFree(gdata.adj2) );
	H_ERR(hipFree(gdata.beg2) );
	H_ERR(hipFree(gdata.buffer) );
	H_ERR(hipFree(gdata.count) );
}



/*
void initDevice(graph* g, int GPU_id,int Part_id){
//cuda memory copy of partAdj and partBegin
	hipSetDevice(0);

	int P=Part_id;
	H_ERR(hipDeviceSynchronize() );

	vertex_t vert_count= g->vert_count;

	vertex_t*	dev_adj;		
	index_t*	dev_begin;	
	index_t*	dev_count;	
	Edge*		buffer0;	
	Edge*		buffer1;	

	index_t EdgeCount = g->partEdgeCount[P];
	vertex_t* Adj   = g->partAdj[P];
	index_t* Begin  = g->partBegin[P];

	H_ERR(hipMalloc(&dev_adj, EdgeCount*sizeof(vertex_t)) );
	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );

	H_ERR(hipMemcpy(dev_adj,    Adj, EdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,  Begin,  (vert_count+1)*sizeof(index_t),  hipMemcpyHostToDevice) );
	
	H_ERR(hipMalloc(&buffer0,    BufferSize*sizeof(Edge)) );
	H_ERR(hipMalloc(&buffer1,    BufferSize*sizeof(Edge)) );
	
	g->gdata[GPU_id].adj	=	dev_adj;
	g->gdata[GPU_id].begin	=	dev_begin;
	g->gdata[GPU_id].count	=	dev_count;
	g->gdata[GPU_id].EdgeBuffer[0]=	buffer0;
	g->gdata[GPU_id].EdgeBuffer[1]=	buffer1;
	g->gdata[GPU_id].partition_id =	P;
	g->gdata[GPU_id].currentBuffer=	0;
	init_count <<<1,max_thd>>>(dev_count);
}

void DeviceCompute(graph* g, int GPU_id, index_t Chunk_id){
	
	int P = g->gdata[GPU_id].partition_id;
//	if(ds_status[P][Chunk_id]!=0) return;	
//	ds_status[P][Chunk_id]=1;
//	if(ds_progress[P]<Chunk_id+1) ds_progress[P] = Chunk_id+1;
	//control
	vertex_t*	dev_adj		=g->gdata[GPU_id].adj;
	index_t*	dev_begin	=g->gdata[GPU_id].begin;
	index_t*	dev_count	=g->gdata[GPU_id].count;
	Edge*		buffer		=g->gdata[GPU_id].EdgeBuffer[g->gdata[GPU_id].currentBuffer];
	g->gdata[GPU_id].currentBuffer	=1-g->gdata[GPU_id].currentBuffer;
	index_t currentBufferSize = BufferSize;
	if(Chunk_id==g->upperEdgeCount/BufferSize){
		currentBufferSize = g->upperEdgeCount % BufferSize;
	}
	init_count <<<1,max_thd>>>(dev_count);
	H_ERR(hipMemcpy(buffer, &g->OrientedEdge[Chunk_id*BufferSize], currentBufferSize*sizeof(Edge), hipMemcpyHostToDevice) );
	H_ERR(hipDeviceSynchronize() );

	warp_binary_kernel<<<max_block,max_thd>>>
	(	buffer,
		dev_adj,
		dev_begin,
		0,
//		GPU_id*256*256/32,
		currentBufferSize,
		dev_count
	);

	//write the result of this chunk back
	H_ERR(hipDeviceSynchronize() );
	index_t tempcount[max_block];
	index_t mycount=0;
	H_ERR(hipMemcpy(tempcount, dev_count, max_block*sizeof(index_t), hipMemcpyDeviceToHost));
	for(int i=0; i<max_block; i++){ mycount += tempcount[i];}
	g->ds_count[P][Chunk_id] = mycount;
//cout<<"chunk count = "<<mycount<<endl;
}

void gpuReduce(graph* g, int GPU_id){
	vertex_t*	dev_adj		=g->gdata[GPU_id].adj;
	index_t*	dev_begin	=g->gdata[GPU_id].begin;
	index_t*	dev_count	=g->gdata[GPU_id].count;
	Edge**		buffer		=g->gdata[GPU_id].EdgeBuffer;
//	H_ERR(hipDeviceSynchronize() );
//	reduce_kernel <<<1,max_thd>>>(dev_count);
//	H_ERR(hipMemcpy(&count[GPU_id], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));
//		thd_count += count[i];
//	count[i] = thd_count;
	H_ERR(hipFree(dev_adj) );
	H_ERR(hipFree(dev_begin) );
	H_ERR(hipFree(dev_count) );
	H_ERR(hipFree(buffer[0]) );
	H_ERR(hipFree(buffer[1]) );
	H_ERR(hipDeviceSynchronize() );
//	cout<<"GPU "<<GPU_id<<" finished"<<endl;
}
*/

